#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <sys/time.h>
#include <iostream>

#ifndef PROBLEM_SIZE
#define PROBLEM_SIZE 200
#endif

#ifndef TIME_STEPS
#define TIME_STEPS 50
#endif


inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}


template <typename T>
bool CompareResult(T *Result, T *Reference, size_t Size) {
  
  T ErrorNorm = 0.0;
  T RefNorm   = 0.0;

  for (unsigned i = 0; i < Size; ++i) {
    double Diff  = Result[i] - Reference[i];
    ErrorNorm  += Diff*Diff;
    RefNorm    += Reference[i]*Reference[i];
  }

  ErrorNorm = std::sqrt(ErrorNorm);
  RefNorm   = std::sqrt(RefNorm);

  std::cout << "Error Norm:  " << ErrorNorm << "\n";
  std::cout << "Ref Norm:    " << RefNorm << "\n";
 
  if(std::abs(RefNorm) < 1e-7) {
    std::cout << "FAIL!\n";
    return false;
  }
  else if((ErrorNorm / RefNorm) > 1e-2) {
    std::cout << "FAIL!\n";
    return false;
  }
  else {
    std::cout << "OK\n";
    return true;
  }
}


void reference(double *u, const double *f, double *g) {

  int   p, n, m;
  int   Iter;
  double r;
  
  const int M = PROBLEM_SIZE;
  const int N = PROBLEM_SIZE;
  const int P = PROBLEM_SIZE;

  const double DT      = 5.0f;
  const double EPSILON = 1.0E-20f;

  double sigma  = 1.00001f;
  double sigma2 = sigma*sigma;
  double lambda = 1.00001f;
  double gamma  = lambda/sigma2;


  double *Temp = new double[M*N*P];

  memcpy(Temp, u, sizeof(double)*M*N*P);
  
#define SQR(x) ((x)*(x))
  
  for(Iter = 1; Iter <= TIME_STEPS; Iter++)
  {
    
    /* Macros for referring to pixel neighbors */
#define CENTER (m+M*(n+N*p))
#define RIGHT  (m+M*(n+N*p)+M)
#define LEFT   (m+M*(n+N*p)-M)
#define DOWN   (m+M*(n+N*p)+1)
#define UP     (m+M*(n+N*p)-1)
#define ZOUT   (m+M*(n+N*p+N))
#define ZIN    (m+M*(n+N*p-N))

    
    /* Approximate g = 1/|grad u| */
    for(p = 1; p < P-1; p++)
      for(n = 1; n < N-1; n++)
        for(m = 1; m < M-1; m++) {
          g[CENTER] = 1.0f/sqrtf( EPSILON
                                  + SQR(u[CENTER] - u[RIGHT])
                                  + SQR(u[CENTER] - u[LEFT])
                                  + SQR(u[CENTER] - u[DOWN])
                                  + SQR(u[CENTER] - u[UP])
                                  + SQR(u[CENTER] - u[ZOUT])
                                  + SQR(u[CENTER] - u[ZIN]));

        }
    
    for(p = 1; p < P-1; p++)
      for(n = 1; n < N-1; n++)
        for(m = 1; m < M-1; m++)
        {
          /* Evaluate r = I1(u*f/sigma^2) / I0(u*f/sigma^2) with
             a cubic rational approximation. */
          r             = u[CENTER]*f[CENTER]/sigma2;
          r             = ( r*(2.38944 + r*(0.950037 + r)) )
            / ( 4.65314 + r*(2.57541 + r*(1.48937 + r)) );
          /* Update u */

          Temp[CENTER] = ( u[CENTER] + DT*(u[RIGHT]*g[RIGHT]
                                          + u[LEFT]*g[LEFT] + u[DOWN]*g[DOWN] + u[UP]*g[UP]
                                          + u[ZOUT]*g[ZOUT] + u[ZIN]*g[ZIN]
                                        + gamma*f[CENTER]*r) ) /
            (1.0 + DT*(g[RIGHT] + g[LEFT]
                       + g[DOWN] + g[UP]
                       + g[ZOUT] + g[ZIN] + gamma));

          
        }


    memcpy(u, Temp, sizeof(double)*M*N*P);
  }


  delete [] Temp;
}


int main() {

  const int Dim_0 = PROBLEM_SIZE;
  const int Dim_1 = PROBLEM_SIZE;
  const int Dim_2 = PROBLEM_SIZE;
  
  double *G = new double[Dim_0*Dim_1*Dim_2];
  double *U = new double[Dim_0*Dim_1*Dim_2];
  double *F = new double[Dim_0*Dim_1*Dim_2];

  srand(time(NULL));
  
  for (int i = 0; i < Dim_0*Dim_1*Dim_2; ++i) {
    G[i] = 0.0f;
    F[i] = (double)rand() / (double)(RAND_MAX+1.0f) * 10.0f;
    U[i] = F[i];
  }
  
  double *RefU = new double[Dim_0*Dim_1*Dim_2];
  double *RefG = new double[Dim_0*Dim_1*Dim_2];

  memcpy(RefU, U, sizeof(double)*Dim_0*Dim_1*Dim_2);
  memcpy(RefG, G, sizeof(double)*Dim_0*Dim_1*Dim_2);


  double RefStart = rtclock();

#ifdef REF_TEST
  reference(RefU, F, RefG);
#endif

  double RefStop = rtclock();


#ifndef REF_TEST
  hipDeviceSynchronize();
#endif

  double Start = rtclock();

#ifndef REF_TEST
  
#pragma overtile begin time_steps:TIME_STEPS block:16,8,8 tile:1,1,1 time:1

  program rician3d is
  grid 3
  field G double inout
  field U double inout
  field F double in

  F[0:0][0:0][0:0] = F[0][0][0]
    
  G[1:1][1:1][1:1] = 
    
    let left   = ((U[0][0][0] - U[0][0][-1])*(U[0][0][0] - U[0][0][-1])) in
    let right  = ((U[0][0][0] - U[0][0][1])*(U[0][0][0] - U[0][0][1])) in
    let top    = ((U[0][0][0] - U[0][-1][0])*(U[0][0][0] - U[0][-1][0])) in
    let bottom = ((U[0][0][0] - U[0][1][0])*(U[0][0][0] - U[0][1][0])) in
    let back   = ((U[0][0][0] - U[-1][0][0])*(U[0][0][0] - U[-1][0][0])) in
    let front  = ((U[0][0][0] - U[1][0][0])*(U[0][0][0] - U[1][0][0])) in
    let epsilon = 1.0e-20 in

      rsqrtf(epsilon + right + left + top + bottom + back + front)

  U[1:1][1:1][1:1] =

    let DT      = 5.0 in
    let sigma   = 1.00001 in
    let sigma2  = sigma*sigma in
    let lambda  = 1.00001 in
    let gamma   = lambda/sigma2 in
    let r_inner = U[0][0][0]*F[0][0][0]/sigma2 in
    let r       = (r_inner*(2.38944 + r_inner*(0.950037+r_inner))) / (4.65314 + r_inner*(2.57541 + r_inner*(1.48937 + r_inner))) in

    let left = U[0][-1][0]*G[0][-1][0] in
    let right  = U[0][1][0]*G[0][1][0] in
    let top    = U[0][0][-1]*G[0][0][-1] in
    let bottom = U[0][0][1]*G[0][0][1] in
    let back  = U[-1][0][0]*G[-1][0][0] in
    let front = U[1][0][0]*G[1][0][0] in

    (U[0][0][0] + DT*(right + left + top + bottom + back + front + gamma*F[0][0][0]*r)) /
      (1.0 + DT*(G[0][0][1] + G[0][0][-1] + G[0][-1][0] + G[0][1][0] + G[-1][0][0] + G[1][0][0] + gamma))

    
#pragma overtile end

#endif

  double Stop = rtclock();

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";
  std::cout << "Ref Elapsed: " << (RefStop - RefStart) << "\n";


  //std::cout << "Check U...\n";
  //CompareResult(U, RefU, Dim_0*Dim_1*Dim_2);

#ifdef REF_TEST
  double GStencils = (Dim_0-2)*(Dim_1-2)*(Dim_2-2)*(double)TIME_STEPS/1e9/(RefStop-RefStart);
  std::cout << "GStencils/sec: " << GStencils << "\n";
#else
  double GStencils = (Dim_0-2)*(Dim_1-2)*(Dim_2-2)*(double)TIME_STEPS/1e9/(Stop-Start);
  std::cout << "GStencils/sec: " << GStencils << "\n";
#endif

  
  delete [] G;
  delete [] U;
  delete [] F;
  delete [] RefG;
  delete [] RefU;

  return 0;
}
