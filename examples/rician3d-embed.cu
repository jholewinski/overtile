#include "hip/hip_runtime.h"

#include <sys/time.h>
#include <iostream>

#ifndef PROBLEM_SIZE
#define PROBLEM_SIZE 200
#endif

#ifndef TIME_STEPS
#define TIME_STEPS 50
#endif


inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}


template <typename T>
bool CompareResult(T *Result, T *Reference, size_t Size) {
  
  T ErrorNorm = 0.0;
  T RefNorm   = 0.0;

  for (unsigned i = 0; i < Size; ++i) {
    float Diff  = Result[i] - Reference[i];
    ErrorNorm  += Diff*Diff;
    RefNorm    += Reference[i]*Reference[i];
  }

  ErrorNorm = std::sqrt(ErrorNorm);
  RefNorm   = std::sqrt(RefNorm);

  std::cout << "Error Norm:  " << ErrorNorm << "\n";
  std::cout << "Ref Norm:    " << RefNorm << "\n";
 
  if(std::abs(RefNorm) < 1e-7) {
    std::cout << "FAIL!\n";
    return false;
  }
  else if((ErrorNorm / RefNorm) > 1e-2) {
    std::cout << "FAIL!\n";
    return false;
  }
  else {
    std::cout << "OK\n";
    return true;
  }
}


void reference(float *u, const float *f, float *g) {

  int   p, n, m;
  int   Iter;
  float r;
  
  const int M = PROBLEM_SIZE;
  const int N = PROBLEM_SIZE;
  const int P = PROBLEM_SIZE;

  const float DT      = 5.0f;
  const float EPSILON = 1.0E-20f;

  float sigma  = 1.00001f;
  float sigma2 = sigma*sigma;
  float lambda = 1.00001f;
  float gamma  = lambda/sigma2;


  float *Temp = new float[M*N*P];

  memcpy(Temp, u, sizeof(float)*M*N*P);
  
#define SQR(x) ((x)*(x))
  
  for(Iter = 1; Iter <= TIME_STEPS; Iter++)
  {
    
    /* Macros for referring to pixel neighbors */
#define CENTER (m+M*(n+N*p))
#define RIGHT  (m+M*(n+N*p)+M)
#define LEFT   (m+M*(n+N*p)-M)
#define DOWN   (m+M*(n+N*p)+1)
#define UP     (m+M*(n+N*p)-1)
#define ZOUT   (m+M*(n+N*p+N))
#define ZIN    (m+M*(n+N*p-N))

    
    /* Approximate g = 1/|grad u| */
    for(p = 1; p < P-1; p++)
      for(n = 1; n < N-1; n++)
        for(m = 1; m < M-1; m++) {
          g[CENTER] = 1.0f/sqrtf( EPSILON
                                  + SQR(u[CENTER] - u[RIGHT])
                                  + SQR(u[CENTER] - u[LEFT])
                                  + SQR(u[CENTER] - u[DOWN])
                                  + SQR(u[CENTER] - u[UP])
                                  + SQR(u[CENTER] - u[ZOUT])
                                  + SQR(u[CENTER] - u[ZIN]));

        }
    
    for(p = 1; p < P-1; p++)
      for(n = 1; n < N-1; n++)
        for(m = 1; m < M-1; m++)
        {
          /* Evaluate r = I1(u*f/sigma^2) / I0(u*f/sigma^2) with
             a cubic rational approximation. */
          r             = u[CENTER]*f[CENTER]/sigma2;
          r             = ( r*(2.38944 + r*(0.950037 + r)) )
            / ( 4.65314 + r*(2.57541 + r*(1.48937 + r)) );
          /* Update u */

          Temp[CENTER] = ( u[CENTER] + DT*(u[RIGHT]*g[RIGHT]
                                          + u[LEFT]*g[LEFT] + u[DOWN]*g[DOWN] + u[UP]*g[UP]
                                          + u[ZOUT]*g[ZOUT] + u[ZIN]*g[ZIN]
                                        + gamma*f[CENTER]*r) ) /
            (1.0 + DT*(g[RIGHT] + g[LEFT]
                       + g[DOWN] + g[UP]
                       + g[ZOUT] + g[ZIN] + gamma));

          
        }


    memcpy(u, Temp, sizeof(float)*M*N*P);
  }


  delete [] Temp;
}


int main() {

  const int Dim_0 = PROBLEM_SIZE;
  const int Dim_1 = PROBLEM_SIZE;
  const int Dim_2 = PROBLEM_SIZE;
  
  float *G = new float[Dim_0*Dim_1*Dim_2];
  float *U = new float[Dim_0*Dim_1*Dim_2];
  float *F = new float[Dim_0*Dim_1*Dim_2];

  srand(time(NULL));
  
  for (int i = 0; i < Dim_0*Dim_1*Dim_2; ++i) {
    G[i] = 0.0f;
    F[i] = (float)rand() / (float)(RAND_MAX+1.0f) * 10.0f;
    U[i] = F[i];
  }
  
  float *RefU = new float[Dim_0*Dim_1*Dim_2];
  float *RefG = new float[Dim_0*Dim_1*Dim_2];

  memcpy(RefU, U, sizeof(float)*Dim_0*Dim_1*Dim_2);
  memcpy(RefG, G, sizeof(float)*Dim_0*Dim_1*Dim_2);


  double RefStart = rtclock();
  
  //reference(RefU, F, RefG);

  double RefStop = rtclock();

  
  hipDeviceSynchronize();

  double Start = rtclock();
  
#pragma overtile begin time_steps:TIME_STEPS block:16,8,8 tile:1,1,1 time:1

  program rician3d is
  grid 3
  field G float inout
  field U float inout
  field F float in

  F[0:0][0:0][0:0] = F[0][0][0]
    
  G[1:1][1:1][1:1] = 
    
    let left   = ((U[0][0][0] - U[0][0][-1])*(U[0][0][0] - U[0][0][-1])) in
    let right  = ((U[0][0][0] - U[0][0][1])*(U[0][0][0] - U[0][0][1])) in
    let top    = ((U[0][0][0] - U[0][-1][0])*(U[0][0][0] - U[0][-1][0])) in
    let bottom = ((U[0][0][0] - U[0][1][0])*(U[0][0][0] - U[0][1][0])) in
    let back   = ((U[0][0][0] - U[-1][0][0])*(U[0][0][0] - U[-1][0][0])) in
    let front  = ((U[0][0][0] - U[1][0][0])*(U[0][0][0] - U[1][0][0])) in
    let epsilon = 1.0e-20 in

      rsqrtf(epsilon + right + left + top + bottom + back + front)

  U[1:1][1:1][1:1] =

    let DT      = 5.0 in
    let sigma   = 1.00001 in
    let sigma2  = sigma*sigma in
    let lambda  = 1.00001 in
    let gamma   = lambda/sigma2 in
    let r_inner = U[0][0][0]*F[0][0][0]/sigma2 in
    let r       = (r_inner*(2.38944 + r_inner*(0.950037+r_inner))) / (4.65314 + r_inner*(2.57541 + r_inner*(1.48937 + r_inner))) in

    let left = U[0][-1][0]*G[0][-1][0] in
    let right  = U[0][1][0]*G[0][1][0] in
    let top    = U[0][0][-1]*G[0][0][-1] in
    let bottom = U[0][0][1]*G[0][0][1] in
    let back  = U[-1][0][0]*G[-1][0][0] in
    let front = U[1][0][0]*G[1][0][0] in

    (U[0][0][0] + DT*(right + left + top + bottom + back + front + gamma*F[0][0][0]*r)) /
      (1.0 + DT*(G[0][0][1] + G[0][0][-1] + G[0][-1][0] + G[0][1][0] + G[-1][0][0] + G[1][0][0] + gamma))

    
#pragma overtile end


  double Stop = rtclock();

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";
  std::cout << "Ref Elapsed: " << (RefStop - RefStart) << "\n";


  std::cout << "Check U...\n";
  CompareResult(U, RefU, Dim_0*Dim_1*Dim_2);



#ifdef PRINT
  //for (int i = 0; i < PROBLEM_SIZE*PROBLEM_SIZE*PROBLEM_SIZE; ++i) {
  //  std::cout << "U: " << U[i] << "  -  Ref U: " << RefU[i] << "\n";
  //}

  for (int i = 1; i < PROBLEM_SIZE-1; ++i) {
    for (int j = 1; j < PROBLEM_SIZE-1; ++j) {
      for (int k = 1; k < PROBLEM_SIZE-1; ++k) {
        std::cout << "U[i][j][k]: " << U[i*PROBLEM_SIZE*PROBLEM_SIZE+j*PROBLEM_SIZE+k] << "  -  RefU[i][j][k]: " << RefU[i*PROBLEM_SIZE*PROBLEM_SIZE+j*PROBLEM_SIZE+k] << "\n";          
      }
    }
  }


  for (int i = 0; i < PROBLEM_SIZE; ++i) {
    for (int j = 0; j < PROBLEM_SIZE; ++j) {
      for (int k = 0; k < PROBLEM_SIZE; ++k) {
        std::cout << "G[i][j][k]: " << G[i*PROBLEM_SIZE*PROBLEM_SIZE+j*PROBLEM_SIZE+k] << "  -  RefG[i][j][k]: " << RefG[i*PROBLEM_SIZE*PROBLEM_SIZE+j*PROBLEM_SIZE+k] << "\n";          
      }
    }
  }

#endif

  
  delete [] G;
  delete [] U;
  delete [] F;
  delete [] RefG;
  delete [] RefU;

  return 0;
}
