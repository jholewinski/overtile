#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <sys/time.h>
#include <iostream>

#ifndef PROBLEM_SIZE
#define PROBLEM_SIZE 200
#endif

#ifndef TIME_STEPS
#define TIME_STEPS 50
#endif


inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}


template <typename T>
bool CompareResult(T *Result, T *Reference, size_t Size) {
  
  T ErrorNorm = 0.0;
  T RefNorm   = 0.0;

  for (unsigned i = 0; i < Size; ++i) {
    float Diff  = Result[i] - Reference[i];
    ErrorNorm  += Diff*Diff;
    RefNorm    += Reference[i]*Reference[i];
  }

  ErrorNorm = std::sqrt(ErrorNorm);
  RefNorm   = std::sqrt(RefNorm);

  std::cout << "Error Norm:  " << ErrorNorm << "\n";
  std::cout << "Ref Norm:    " << RefNorm << "\n";
 
  if(std::abs(RefNorm) < 1e-7) {
    std::cout << "FAIL!\n";
    return false;
  }
  else if((ErrorNorm / RefNorm) > 1e-2) {
    std::cout << "FAIL!\n";
    return false;
  }
  else {
    std::cout << "OK\n";
    return true;
  }
}


void reference(float *u, const float *f, float *g) {

  int   p, n, m;
  int   Iter;
  float r;
  
  const int M = PROBLEM_SIZE;
  const int N = PROBLEM_SIZE;
  const int P = PROBLEM_SIZE;

  const float DT      = 5.0f;
  const float EPSILON = 1.0E-20f;

  float sigma  = 1.00001f;
  float sigma2 = sigma*sigma;
  float lambda = 1.00001f;
  float gamma  = lambda/sigma2;


  float *Temp = new float[M*N*P];

  memcpy(Temp, u, sizeof(float)*M*N*P);
  
#define SQR(x) ((x)*(x))
  
  for(Iter = 1; Iter <= TIME_STEPS; Iter++)
  {
    
    /* Macros for referring to pixel neighbors */
#define CENTER (m+M*(n+N*p))
#define RIGHT  (m+M*(n+N*p)+M)
#define LEFT   (m+M*(n+N*p)-M)
#define DOWN   (m+M*(n+N*p)+1)
#define UP     (m+M*(n+N*p)-1)
#define ZOUT   (m+M*(n+N*p+N))
#define ZIN    (m+M*(n+N*p-N))

    
    /* Approximate g = 1/|grad u| */
    for(p = 1; p < P-1; p++)
      for(n = 1; n < N-1; n++)
        for(m = 1; m < M-1; m++) {
          g[CENTER] = 1.0f/sqrtf( EPSILON
                                  + SQR(u[CENTER] - u[RIGHT])
                                  + SQR(u[CENTER] - u[LEFT])
                                  + SQR(u[CENTER] - u[DOWN])
                                  + SQR(u[CENTER] - u[UP])
                                  + SQR(u[CENTER] - u[ZOUT])
                                  + SQR(u[CENTER] - u[ZIN]));

        }
    
    for(p = 1; p < P-1; p++)
      for(n = 1; n < N-1; n++)
        for(m = 1; m < M-1; m++)
        {
          /* Evaluate r = I1(u*f/sigma^2) / I0(u*f/sigma^2) with
             a cubic rational approximation. */
          r             = u[CENTER]*f[CENTER]/sigma2;
          r             = ( r*(2.38944 + r*(0.950037 + r)) )
            / ( 4.65314 + r*(2.57541 + r*(1.48937 + r)) );
          /* Update u */

          Temp[CENTER] = ( u[CENTER] + DT*(u[RIGHT]*g[RIGHT]
                                          + u[LEFT]*g[LEFT] + u[DOWN]*g[DOWN] + u[UP]*g[UP]
                                          + u[ZOUT]*g[ZOUT] + u[ZIN]*g[ZIN]
                                        + gamma*f[CENTER]*r) ) /
            (1.0 + DT*(g[RIGHT] + g[LEFT]
                       + g[DOWN] + g[UP]
                       + g[ZOUT] + g[ZIN] + gamma));

          
        }


    memcpy(u, Temp, sizeof(float)*M*N*P);
  }


  delete [] Temp;
}


int main() {

  const int Dim_0 = PROBLEM_SIZE;
  const int Dim_1 = PROBLEM_SIZE;
  const int Dim_2 = PROBLEM_SIZE;
  
  float *U = new float[Dim_0*Dim_1*Dim_2];
  float *F = new float[Dim_0*Dim_1*Dim_2];

  srand(time(NULL));
  
  for (int i = 0; i < Dim_0*Dim_1*Dim_2; ++i) {
    F[i] = (float)rand() / (float)(RAND_MAX+1.0f) * 10.0f;
    U[i] = F[i];
  }
  
  float *RefU = new float[Dim_0*Dim_1*Dim_2];

  memcpy(RefU, U, sizeof(float)*Dim_0*Dim_1*Dim_2);


  double RefStart = rtclock();

#ifdef REF_TEST
  reference(RefU, F, RefG);
#endif

  double RefStop = rtclock();


#ifndef REF_TEST
  hipDeviceSynchronize();
#endif

  double Start = rtclock();

#ifndef REF_TEST
  
#pragma overtile begin time_steps:TIME_STEPS block:12,8,8 tile:1,1,2 time:1

  program rician3d is
  grid 3
  field U float inout
  field F float in

  U[1:1][1:1][1:1] =

    let DT      = 5.0 in
    let sigma   = 1.00001 in
    let sigma2  = sigma*sigma in
    let lambda  = 1.00001 in
    let gamma   = lambda/sigma2 in
    let r_inner = U[0][0][0]*F[0][0][0]/sigma2 in
    let r       = (r_inner*(2.38944 + r_inner*(0.950037+r_inner))) / (4.65314 + r_inner*(2.57541 + r_inner*(1.48937 + r_inner))) in

    let epsilon = 1.0e-20 in


    let left_1_0_0   = ((U[1][0][0] - U[1][0][-1])*(U[1][0][0] - U[1][0][-1])) in
    let right_1_0_0  = ((U[1][0][0] - U[1][0][1])*(U[1][0][0] - U[1][0][1])) in
    let top_1_0_0    = ((U[1][0][0] - U[1][-1][0])*(U[1][0][0] - U[1][-1][0])) in
    let bottom_1_0_0 = ((U[1][0][0] - U[1][1][0])*(U[1][0][0] - U[1][1][0])) in
    let back_1_0_0   = ((U[1][0][0] - U[0][0][0])*(U[1][0][0] - U[0][0][0])) in
    let front_1_0_0  = ((U[1][0][0] - U[2][0][0])*(U[1][0][0] - U[2][0][0])) in
    let g_1_0_0      = rsqrtf(epsilon + right_1_0_0 + left_1_0_0 + top_1_0_0 + bottom_1_0_0 + back_1_0_0 + front_1_0_0) in

    let left_m1_0_0   = ((U[-1][0][0] - U[-1][0][-1])*(U[-1][0][0] - U[-1][0][-1])) in
    let right_m1_0_0  = ((U[-1][0][0] - U[-1][0][1])*(U[-1][0][0] - U[-1][0][1])) in
    let top_m1_0_0    = ((U[-1][0][0] - U[-1][-1][0])*(U[-1][0][0] - U[-1][-1][0])) in
    let bottom_m1_0_0 = ((U[-1][0][0] - U[-1][1][0])*(U[-1][0][0] - U[-1][1][0])) in
    let back_m1_0_0   = ((U[-1][0][0] - U[-2][0][0])*(U[-1][0][0] - U[-2][0][0])) in
    let front_m1_0_0  = ((U[-1][0][0] - U[0][0][0])*(U[-1][0][0] - U[0][0][0])) in
    let g_m1_0_0      = rsqrtf(epsilon + right_m1_0_0 + left_m1_0_0 + top_m1_0_0 + bottom_m1_0_0 + back_m1_0_0 + front_m1_0_0) in

    let left_0_1_0   = ((U[0][1][0] - U[0][1][-1])*(U[0][1][0] - U[0][1][-1])) in
    let right_0_1_0  = ((U[0][1][0] - U[0][1][1])*(U[0][1][0] - U[0][1][1])) in
    let top_0_1_0    = ((U[0][1][0] - U[0][0][0])*(U[0][1][0] - U[0][0][0])) in
    let bottom_0_1_0 = ((U[0][1][0] - U[0][2][0])*(U[0][1][0] - U[0][2][0])) in
    let back_0_1_0   = ((U[0][1][0] - U[-1][1][0])*(U[0][1][0] - U[-1][1][0])) in
    let front_0_1_0  = ((U[0][1][0] - U[1][1][0])*(U[0][1][0] - U[1][1][0])) in
    let g_0_1_0      = rsqrtf(epsilon + right_0_1_0 + left_0_1_0 + top_0_1_0 + bottom_0_1_0 + back_0_1_0 + front_0_1_0) in

    let left_0_m1_0   = ((U[0][-1][0] - U[0][-1][-1])*(U[0][-1][0] - U[0][-1][-1])) in
    let right_0_m1_0  = ((U[0][-1][0] - U[0][-1][1])*(U[0][-1][0] - U[0][-1][1])) in
    let top_0_m1_0    = ((U[0][-1][0] - U[0][-2][0])*(U[0][-1][0] - U[0][-2][0])) in
    let bottom_0_m1_0 = ((U[0][-1][0] - U[0][0][0])*(U[0][-1][0] - U[0][0][0])) in
    let back_0_m1_0   = ((U[0][-1][0] - U[-1][-1][0])*(U[0][-1][0] - U[-1][-1][0])) in
    let front_0_m1_0  = ((U[0][-1][0] - U[1][-1][0])*(U[0][-1][0] - U[1][-1][0])) in
    let g_0_m1_0      = rsqrtf(epsilon + right_0_m1_0 + left_0_m1_0 + top_0_m1_0 + bottom_0_m1_0 + back_0_m1_0 + front_0_m1_0) in

    let left_0_0_1   = ((U[0][0][1] - U[0][0][0])*(U[0][0][1] - U[0][0][0])) in
    let right_0_0_1  = ((U[0][0][1] - U[0][0][2])*(U[0][0][1] - U[0][0][2])) in
    let top_0_0_1    = ((U[0][0][1] - U[0][-1][1])*(U[0][0][1] - U[0][-1][1])) in
    let bottom_0_0_1 = ((U[0][0][1] - U[0][1][1])*(U[0][0][1] - U[0][1][1])) in
    let back_0_0_1   = ((U[0][0][1] - U[-1][0][1])*(U[0][0][1] - U[-1][0][1])) in
    let front_0_0_1  = ((U[0][0][1] - U[1][0][1])*(U[0][0][1] - U[1][0][1])) in
    let g_0_0_1      = rsqrtf(epsilon + right_0_0_1 + left_0_0_1 + top_0_0_1 + bottom_0_0_1 + back_0_0_1 + front_0_0_1) in

    let left_0_0_m1   = ((U[0][0][-1] - U[0][0][-2])*(U[0][0][-1] - U[0][0][-2])) in
    let right_0_0_m1  = ((U[0][0][-1] - U[0][0][0])*(U[0][0][-1] - U[0][0][0])) in
    let top_0_0_m1    = ((U[0][0][-1] - U[0][-1][-1])*(U[0][0][-1] - U[0][-1][1])) in
    let bottom_0_0_m1 = ((U[0][0][-1] - U[0][1][-1])*(U[0][0][-1] - U[0][1][1])) in
    let back_0_0_m1   = ((U[0][0][-1] - U[-1][0][-1])*(U[0][0][-1] - U[-1][0][1])) in
    let front_0_0_m1  = ((U[0][0][-1] - U[1][0][-1])*(U[0][0][-1] - U[1][0][1])) in
    let g_0_0_m1      = rsqrtf(epsilon + right_0_0_m1 + left_0_0_m1 + top_0_0_m1 + bottom_0_0_m1 + back_0_0_m1 + front_0_0_m1) in



    
    let left   = U[0][-1][0]*g_0_m1_0 in
    let right  = U[0][1][0]*g_0_1_0 in
    let top    = U[0][0][-1]*g_0_0_m1 in
    let bottom = U[0][0][1]*g_0_0_1 in
    let back   = U[-1][0][0]*g_m1_0_0 in
    let front  = U[1][0][0]*g_1_0_0 in

    (U[0][0][0] + DT*(right + left + top + bottom + back + front + gamma*F[0][0][0]*r)) /
      (1.0 + DT*(g_0_0_1 + g_0_0_m1 + g_0_m1_0 + g_0_1_0 + g_m1_0_0 + g_1_0_0 + gamma))

    
#pragma overtile end

    /*
          let left_0_0_0 = ((U[0][0][0] - U[0][0][-1])*(U[0][0][0] - U[0][0][-1])) in
    let right_0_0_0      = ((U[0][0][0] - U[0][0][1])*(U[0][0][0] - U[0][0][1])) in
    let top_0_0_0        = ((U[0][0][0] - U[0][-1][0])*(U[0][0][0] - U[0][-1][0])) in
    let bottom_0_0_0     = ((U[0][0][0] - U[0][1][0])*(U[0][0][0] - U[0][1][0])) in
    let back_0_0_0       = ((U[0][0][0] - U[-1][0][0])*(U[0][0][0] - U[-1][0][0])) in
    let front_0_0_0      = ((U[0][0][0] - U[1][0][0])*(U[0][0][0] - U[1][0][0])) in
    let g_0_0_0          = rsqrtf(epsilon + right_0_0_0 + left_0_0_0 + top_0_0_0 + bottom_0_0_0 + back_0_0_0 + front_0_0_0) in
    */
    
#endif

  double Stop = rtclock();

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";
  std::cout << "Ref Elapsed: " << (RefStop - RefStart) << "\n";


  //std::cout << "Check U...\n";
  //CompareResult(U, RefU, Dim_0*Dim_1*Dim_2);

#ifdef REF_TEST
  double GStencils = (Dim_0-2)*(Dim_1-2)*(Dim_2-2)*(double)TIME_STEPS/1e9/(RefStop-RefStart);
  std::cout << "GStencils/sec: " << GStencils << "\n";
#else
  double GStencils = (Dim_0-2)*(Dim_1-2)*(Dim_2-2)*(double)TIME_STEPS/1e9/(Stop-Start);
  std::cout << "GStencils/sec: " << GStencils << "\n";
#endif

  
  delete [] U;
  delete [] F;
  delete [] RefU;

  return 0;
}
