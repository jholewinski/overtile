#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <sys/time.h>
#include <iostream>

#ifndef PROBLEM_SIZE
#define PROBLEM_SIZE 6000
#endif

#ifndef TIME_STEPS
#define TIME_STEPS 100
#endif


inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}


template <typename T>
bool CompareResult(T *Result, T *Reference, size_t Size) {
  
  T ErrorNorm = 0.0;
  T RefNorm   = 0.0;

  for (unsigned i = 0; i < Size; ++i) {
    float Diff  = Result[i] - Reference[i];
    ErrorNorm  += Diff*Diff;
    RefNorm    += Reference[i]*Reference[i];
  }

  ErrorNorm = std::sqrt(ErrorNorm);
  RefNorm   = std::sqrt(RefNorm);

  std::cout << "Error Norm:  " << ErrorNorm << "\n";
  std::cout << "Ref Norm:    " << RefNorm << "\n";
 
  if(std::abs(RefNorm) < 1e-7) {
    std::cout << "FAIL!\n";
    return false;
  }
  else if((ErrorNorm / RefNorm) > 1e-2) {
    std::cout << "FAIL!\n";
    return false;
  }
  else {
    std::cout << "OK\n";
    return true;
  }
}


void reference(float *u, const float *f, const float *V) {

  int   m, n;
  int   Iter;
  float r;
  
  const int M = PROBLEM_SIZE;
  const int N = PROBLEM_SIZE;

  float *Temp = new float[N*M];

  const float epi2  = 1e-8f;
  const float alpha = 1.001;
  
#define REF(i,j) ((i)*M+(j))
#define SQR(x)   ((x)*(x))
  
  for (int t = 0; t < TIME_STEPS; ++t) {
    for (int i = 1; i < N-1; ++i) {
      for (int j = 1; j < M-1; ++j) {

        float c1 = u[REF(i,j)] / sqrtf(epi2 + SQR(u[REF(i+1,j)]-u[REF(i,j)]) + SQR(u[REF(i,j+1)]-u[REF(i,j)])) / V[REF(i,j)];
        float c2 = u[REF(i,j)] / sqrtf(epi2 + SQR(u[REF(i,j)]-u[REF(i-1,j)]) + SQR(u[REF(i-1,j+1)]-u[REF(i-1,j)])) / V[REF(i,j)];
        float c3 = u[REF(i,j)] / sqrtf(epi2 + SQR(u[REF(i+1,j)]-u[REF(i,j)]) + SQR(u[REF(i,j+1)]-u[REF(i,j)])) / V[REF(i,j)];
        float c4 = u[REF(i,j)] / sqrtf(epi2 + SQR(u[REF(i+1,j-1)]-u[REF(i,j-1)]) + SQR(u[REF(i,j)]-u[REF(i,j-1)])) / V[REF(i,j)];

        
        Temp[REF(i,j)] = 1.0f / (alpha+c1+c2+c3+c4)*(alpha*f[REF(i,j)]+c1*u[REF(i+1,j)]+c2*u[REF(i-1,j)]+c3*u[REF(i,j+1)]+c4*u[REF(i,j-1)]);
      }
    }

    memcpy(u, Temp, sizeof(float)*N*M);
  }

  delete [] Temp;
}


int main() {

  const int Dim_0 = PROBLEM_SIZE;
  const int Dim_1 = PROBLEM_SIZE;
  
  float *V = new float[Dim_0*Dim_1];
  float *u = new float[Dim_0*Dim_1];
  float *f = new float[Dim_0*Dim_1];

  srand(time(NULL));
  
  for (int i = 0; i < Dim_0*Dim_1; ++i) {
    V[i] = (float)rand() / (float)(RAND_MAX+1.0f) * 10.0f;
    f[i] = (float)rand() / (float)(RAND_MAX+1.0f) * 10.0f;
    u[i] = (float)rand() / (float)(RAND_MAX+1.0f) * 10.0f;
  }
  
  float *RefU = new float[Dim_0*Dim_1];
  float *RefF = new float[Dim_0*Dim_1];
  float *RefV = new float[Dim_0*Dim_1];

  memcpy(RefU, u, sizeof(float)*Dim_0*Dim_1);
  memcpy(RefF, f, sizeof(float)*Dim_0*Dim_1);
  memcpy(RefV, V, sizeof(float)*Dim_0*Dim_1);


  double RefStart = rtclock();

#ifdef REF_TEST
  reference(RefU, RefF, RefV);
#endif
  
  double RefStop = rtclock();

#ifndef REF_TEST  
  hipDeviceSynchronize();
#endif

  double Start = rtclock();

#ifndef REF_TEST
  
#pragma overtile begin time_steps:TIME_STEPS block:32,8 tile:1,6 time:1

program tv2d is

  grid 2

  field u float inout
  field V float in
  field f float in

  u[1:1][1:1] = 
  let epi2    = 1e-8 in
  let alpha   = 1.001 in
  let c1      = u[0][0] / sqrtf(epi2 + SQR(u[1][0]-u[0][0]) + SQR(u[0][1]-u[0][0])) / V[0][0] in
  let c2      = u[0][0] / sqrtf(epi2 + SQR(u[0][0]-u[-1][0]) + SQR(u[-1][1]-u[-1][0])) / V[0][0] in
  let c3      = u[0][0] / sqrtf(epi2 + SQR(u[1][0]-u[0][0]) + SQR(u[0][1]-u[0][0])) / V[0][0] in
  let c4      = u[0][0] / sqrtf(epi2 + SQR(u[1][-1]-u[0][-1]) + SQR(u[0][0]-u[0][-1])) / V[0][0] in
       
    1.0 / (alpha+c1+c2+c3+c4)*(alpha*f[0][0]+c1*u[1][0]+c2*u[-1][0]+c3*u[0][1]+c4*u[0][-1])

    
#pragma overtile end

#endif

  double Stop = rtclock();

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";
  std::cout << "Ref Elapsed: " << (RefStop - RefStart) << "\n";


  std::cout << "Check U...\n";
  CompareResult(u, RefU, Dim_0*Dim_1);


#ifdef REF_TEST
  double GStencils = (Dim_0-2)*(Dim_1-2)*(double)TIME_STEPS/1e9/(RefStop-RefStart);
  std::cout << "GStencils/sec: " << GStencils << "\n";
#else
  double GStencils = (Dim_0-2)*(Dim_1-2)*(double)TIME_STEPS/1e9/(Stop-Start);
  std::cout << "GStencils/sec: " << GStencils << "\n";
#endif


  
  delete [] V;
  delete [] u;
  delete [] f;
  delete [] RefF;
  delete [] RefU;
  delete [] RefV;

  return 0;
}
