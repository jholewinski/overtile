#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <sys/time.h>
#include <iostream>

#ifndef PROBLEM_SIZE
#define PROBLEM_SIZE 200
#endif

#ifndef TIME_STEPS
#define TIME_STEPS 50
#endif


inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}


template <typename T>
bool CompareResult(T *Result, T *Reference, size_t Size) {
  
  T ErrorNorm = 0.0;
  T RefNorm   = 0.0;

  for (unsigned i = 0; i < Size; ++i) {
    double Diff  = Result[i] - Reference[i];
    ErrorNorm  += Diff*Diff;
    RefNorm    += Reference[i]*Reference[i];
  }

  ErrorNorm = std::sqrt(ErrorNorm);
  RefNorm   = std::sqrt(RefNorm);

  std::cout << "Error Norm:  " << ErrorNorm << "\n";
  std::cout << "Ref Norm:    " << RefNorm << "\n";
 
  if(std::abs(RefNorm) < 1e-7) {
    std::cout << "FAIL!\n";
    return false;
  }
  else if((ErrorNorm / RefNorm) > 1e-2) {
    std::cout << "FAIL!\n";
    return false;
  }
  else {
    std::cout << "OK\n";
    return true;
  }
}


void reference(double *u, const double *f, double *g) {

  int   p, n, m;
  int   Iter;
  double r;
  
  const int M = PROBLEM_SIZE;
  const int N = PROBLEM_SIZE;
  const int P = PROBLEM_SIZE;

  const double DT      = 5.0f;
  const double EPSILON = 1.0E-20f;

  double sigma  = 1.00001f;
  double sigma2 = sigma*sigma;
  double lambda = 1.00001f;
  double gamma  = lambda/sigma2;


  double *Temp = new double[M*N*P];

  memcpy(Temp, u, sizeof(double)*M*N*P);
  
#define SQR(x) ((x)*(x))
  
  for(Iter = 1; Iter <= TIME_STEPS; Iter++)
  {
    
    /* Macros for referring to pixel neighbors */
#define CENTER (m+M*(n+N*p))
#define RIGHT  (m+M*(n+N*p)+M)
#define LEFT   (m+M*(n+N*p)-M)
#define DOWN   (m+M*(n+N*p)+1)
#define UP     (m+M*(n+N*p)-1)
#define ZOUT   (m+M*(n+N*p+N))
#define ZIN    (m+M*(n+N*p-N))

    
    /* Approximate g = 1/|grad u| */
    for(p = 1; p < P-1; p++)
      for(n = 1; n < N-1; n++)
        for(m = 1; m < M-1; m++) {
          g[CENTER] = 1.0f/sqrtf( EPSILON
                                  + SQR(u[CENTER] - u[RIGHT])
                                  + SQR(u[CENTER] - u[LEFT])
                                  + SQR(u[CENTER] - u[DOWN])
                                  + SQR(u[CENTER] - u[UP])
                                  + SQR(u[CENTER] - u[ZOUT])
                                  + SQR(u[CENTER] - u[ZIN]));

        }
    
    for(p = 1; p < P-1; p++)
      for(n = 1; n < N-1; n++)
        for(m = 1; m < M-1; m++)
        {
          /* Evaluate r = I1(u*f/sigma^2) / I0(u*f/sigma^2) with
             a cubic rational approximation. */
          r             = u[CENTER]*f[CENTER]/sigma2;
          r             = ( r*(2.38944 + r*(0.950037 + r)) )
            / ( 4.65314 + r*(2.57541 + r*(1.48937 + r)) );
          /* Update u */

          Temp[CENTER] = ( u[CENTER] + DT*(u[RIGHT]*g[RIGHT]
                                          + u[LEFT]*g[LEFT] + u[DOWN]*g[DOWN] + u[UP]*g[UP]
                                          + u[ZOUT]*g[ZOUT] + u[ZIN]*g[ZIN]
                                        + gamma*f[CENTER]*r) ) /
            (1.0 + DT*(g[RIGHT] + g[LEFT]
                       + g[DOWN] + g[UP]
                       + g[ZOUT] + g[ZIN] + gamma));

          
        }


    memcpy(u, Temp, sizeof(double)*M*N*P);
  }


  delete [] Temp;
}


int main() {

  const int Dim_0 = PROBLEM_SIZE;
  const int Dim_1 = PROBLEM_SIZE;
  const int Dim_2 = PROBLEM_SIZE;
  
  double *U = new double[Dim_0*Dim_1*Dim_2];
  double *F = new double[Dim_0*Dim_1*Dim_2];

  srand(time(NULL));
  
  for (int i = 0; i < Dim_0*Dim_1*Dim_2; ++i) {
    F[i] = (double)rand() / (double)(RAND_MAX+1.0f) * 10.0f;
    U[i] = F[i];
  }
  
  double *RefU = new double[Dim_0*Dim_1*Dim_2];

  memcpy(RefU, U, sizeof(double)*Dim_0*Dim_1*Dim_2);


  double RefStart = rtclock();

#ifdef REF_TEST
  reference(RefU, F, RefG);
#endif

  double RefStop = rtclock();


#ifndef REF_TEST
  hipDeviceSynchronize();
#endif

  double Start = rtclock();

#ifndef REF_TEST
  
#pragma overtile begin time_steps:TIME_STEPS block:8,8,8 tile:1,1,1 time:1

  program rician3d is
  grid 3
  field U double inout
  field F double in

  U[1:1][1:1][1:1] =

    let DT      = 5.0 in
    let sigma   = 1.00001 in
    let sigma2  = sigma*sigma in
    let lambda  = 1.00001 in
    let gamma   = lambda/sigma2 in
    let r_inner = U[0][0][0]*F[0][0][0]/sigma2 in
    let r       = (r_inner*(2.38944 + r_inner*(0.950037+r_inner))) / (4.65314 + r_inner*(2.57541 + r_inner*(1.48937 + r_inner))) in

    let epsilon = 1.0e-20 in


    let left_1_0_0   = ((U[1][0][0] - U[1][0][-1])*(U[1][0][0] - U[1][0][-1])) in
    let right_1_0_0  = ((U[1][0][0] - U[1][0][1])*(U[1][0][0] - U[1][0][1])) in
    let top_1_0_0    = ((U[1][0][0] - U[1][-1][0])*(U[1][0][0] - U[1][-1][0])) in
    let bottom_1_0_0 = ((U[1][0][0] - U[1][1][0])*(U[1][0][0] - U[1][1][0])) in
    let back_1_0_0   = ((U[1][0][0] - U[0][0][0])*(U[1][0][0] - U[0][0][0])) in
    let front_1_0_0  = ((U[1][0][0] - U[2][0][0])*(U[1][0][0] - U[2][0][0])) in
    let g_1_0_0      = rsqrtf(epsilon + right_1_0_0 + left_1_0_0 + top_1_0_0 + bottom_1_0_0 + back_1_0_0 + front_1_0_0) in

    let left_m1_0_0   = ((U[-1][0][0] - U[-1][0][-1])*(U[-1][0][0] - U[-1][0][-1])) in
    let right_m1_0_0  = ((U[-1][0][0] - U[-1][0][1])*(U[-1][0][0] - U[-1][0][1])) in
    let top_m1_0_0    = ((U[-1][0][0] - U[-1][-1][0])*(U[-1][0][0] - U[-1][-1][0])) in
    let bottom_m1_0_0 = ((U[-1][0][0] - U[-1][1][0])*(U[-1][0][0] - U[-1][1][0])) in
    let back_m1_0_0   = ((U[-1][0][0] - U[-2][0][0])*(U[-1][0][0] - U[-2][0][0])) in
    let front_m1_0_0  = ((U[-1][0][0] - U[0][0][0])*(U[-1][0][0] - U[0][0][0])) in
    let g_m1_0_0      = rsqrtf(epsilon + right_m1_0_0 + left_m1_0_0 + top_m1_0_0 + bottom_m1_0_0 + back_m1_0_0 + front_m1_0_0) in

    let left_0_1_0   = ((U[0][1][0] - U[0][1][-1])*(U[0][1][0] - U[0][1][-1])) in
    let right_0_1_0  = ((U[0][1][0] - U[0][1][1])*(U[0][1][0] - U[0][1][1])) in
    let top_0_1_0    = ((U[0][1][0] - U[0][0][0])*(U[0][1][0] - U[0][0][0])) in
    let bottom_0_1_0 = ((U[0][1][0] - U[0][2][0])*(U[0][1][0] - U[0][2][0])) in
    let back_0_1_0   = ((U[0][1][0] - U[-1][1][0])*(U[0][1][0] - U[-1][1][0])) in
    let front_0_1_0  = ((U[0][1][0] - U[1][1][0])*(U[0][1][0] - U[1][1][0])) in
    let g_0_1_0      = rsqrtf(epsilon + right_0_1_0 + left_0_1_0 + top_0_1_0 + bottom_0_1_0 + back_0_1_0 + front_0_1_0) in

    let left_0_m1_0   = ((U[0][-1][0] - U[0][-1][-1])*(U[0][-1][0] - U[0][-1][-1])) in
    let right_0_m1_0  = ((U[0][-1][0] - U[0][-1][1])*(U[0][-1][0] - U[0][-1][1])) in
    let top_0_m1_0    = ((U[0][-1][0] - U[0][-2][0])*(U[0][-1][0] - U[0][-2][0])) in
    let bottom_0_m1_0 = ((U[0][-1][0] - U[0][0][0])*(U[0][-1][0] - U[0][0][0])) in
    let back_0_m1_0   = ((U[0][-1][0] - U[-1][-1][0])*(U[0][-1][0] - U[-1][-1][0])) in
    let front_0_m1_0  = ((U[0][-1][0] - U[1][-1][0])*(U[0][-1][0] - U[1][-1][0])) in
    let g_0_m1_0      = rsqrtf(epsilon + right_0_m1_0 + left_0_m1_0 + top_0_m1_0 + bottom_0_m1_0 + back_0_m1_0 + front_0_m1_0) in

    let left_0_0_1   = ((U[0][0][1] - U[0][0][0])*(U[0][0][1] - U[0][0][0])) in
    let right_0_0_1  = ((U[0][0][1] - U[0][0][2])*(U[0][0][1] - U[0][0][2])) in
    let top_0_0_1    = ((U[0][0][1] - U[0][-1][1])*(U[0][0][1] - U[0][-1][1])) in
    let bottom_0_0_1 = ((U[0][0][1] - U[0][1][1])*(U[0][0][1] - U[0][1][1])) in
    let back_0_0_1   = ((U[0][0][1] - U[-1][0][1])*(U[0][0][1] - U[-1][0][1])) in
    let front_0_0_1  = ((U[0][0][1] - U[1][0][1])*(U[0][0][1] - U[1][0][1])) in
    let g_0_0_1      = rsqrtf(epsilon + right_0_0_1 + left_0_0_1 + top_0_0_1 + bottom_0_0_1 + back_0_0_1 + front_0_0_1) in

    let left_0_0_m1   = ((U[0][0][-1] - U[0][0][-2])*(U[0][0][-1] - U[0][0][-2])) in
    let right_0_0_m1  = ((U[0][0][-1] - U[0][0][0])*(U[0][0][-1] - U[0][0][0])) in
    let top_0_0_m1    = ((U[0][0][-1] - U[0][-1][-1])*(U[0][0][-1] - U[0][-1][1])) in
    let bottom_0_0_m1 = ((U[0][0][-1] - U[0][1][-1])*(U[0][0][-1] - U[0][1][1])) in
    let back_0_0_m1   = ((U[0][0][-1] - U[-1][0][-1])*(U[0][0][-1] - U[-1][0][1])) in
    let front_0_0_m1  = ((U[0][0][-1] - U[1][0][-1])*(U[0][0][-1] - U[1][0][1])) in
    let g_0_0_m1      = rsqrtf(epsilon + right_0_0_m1 + left_0_0_m1 + top_0_0_m1 + bottom_0_0_m1 + back_0_0_m1 + front_0_0_m1) in



    
    let left   = U[0][-1][0]*g_0_m1_0 in
    let right  = U[0][1][0]*g_0_1_0 in
    let top    = U[0][0][-1]*g_0_0_m1 in
    let bottom = U[0][0][1]*g_0_0_1 in
    let back   = U[-1][0][0]*g_m1_0_0 in
    let front  = U[1][0][0]*g_1_0_0 in

    (U[0][0][0] + DT*(right + left + top + bottom + back + front + gamma*F[0][0][0]*r)) /
      (1.0 + DT*(g_0_0_1 + g_0_0_m1 + g_0_m1_0 + g_0_1_0 + g_m1_0_0 + g_1_0_0 + gamma))

    
#pragma overtile end

    /*
          let left_0_0_0 = ((U[0][0][0] - U[0][0][-1])*(U[0][0][0] - U[0][0][-1])) in
    let right_0_0_0      = ((U[0][0][0] - U[0][0][1])*(U[0][0][0] - U[0][0][1])) in
    let top_0_0_0        = ((U[0][0][0] - U[0][-1][0])*(U[0][0][0] - U[0][-1][0])) in
    let bottom_0_0_0     = ((U[0][0][0] - U[0][1][0])*(U[0][0][0] - U[0][1][0])) in
    let back_0_0_0       = ((U[0][0][0] - U[-1][0][0])*(U[0][0][0] - U[-1][0][0])) in
    let front_0_0_0      = ((U[0][0][0] - U[1][0][0])*(U[0][0][0] - U[1][0][0])) in
    let g_0_0_0          = rsqrtf(epsilon + right_0_0_0 + left_0_0_0 + top_0_0_0 + bottom_0_0_0 + back_0_0_0 + front_0_0_0) in
    */
    
#endif

  double Stop = rtclock();

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";
  std::cout << "Ref Elapsed: " << (RefStop - RefStart) << "\n";


  //std::cout << "Check U...\n";
  //CompareResult(U, RefU, Dim_0*Dim_1*Dim_2);

#ifdef REF_TEST
  double GStencils = (Dim_0-2)*(Dim_1-2)*(Dim_2-2)*(double)TIME_STEPS/1e9/(RefStop-RefStart);
  std::cout << "GStencils/sec: " << GStencils << "\n";
#else
  double GStencils = (Dim_0-2)*(Dim_1-2)*(Dim_2-2)*(double)TIME_STEPS/1e9/(Stop-Start);
  std::cout << "GStencils/sec: " << GStencils << "\n";
#endif

  
  delete [] U;
  delete [] F;
  delete [] RefU;

  return 0;
}
