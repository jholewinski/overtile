
#include <sys/time.h>
#include <iostream>

#ifndef PROBLEM_SIZE
#define PROBLEM_SIZE 200
#endif

#ifndef TIME_STEPS
#define TIME_STEPS 100
#endif


inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}





int main() {

  const int Dim_0 = PROBLEM_SIZE;
  const int Dim_1 = PROBLEM_SIZE;
  const int Dim_2 = PROBLEM_SIZE;
  
  float *Ex = new float[Dim_0*Dim_1*Dim_2];
  float *Ey = new float[Dim_0*Dim_1*Dim_2];
  float *Ez = new float[Dim_0*Dim_1*Dim_2];
  float *Hz = new float[Dim_0*Dim_1*Dim_2];

  hipDeviceSynchronize();

  double Start = rtclock();
  
#pragma overtile begin time_steps:TIME_STEPS block:16,4,4 tile:1,1,1 time:1

  program fdtd3d is
  grid 3
  field Ex float inout
  field Ey float inout
  field Ez float inout
  field Hz float inout
    
    Ey[1:0][0:0][0:0] = Ey[0][0][0] - 0.5*(Hz[0][0][0] - Hz[-1][0][0])
    
    Ex[0:0][1:0][0:0] = Ex[0][0][0] - 0.5*(Hz[0][0][0] - Hz[0][-1][0])

    Ez[0:0][0:0][1:0] = Ez[0][0][0] - 0.5*(Hz[0][0][0] - Hz[0][0][-1])

    Hz[0:1][0:1][0:1] = Hz[0][0][0] - 0.7*(Ex[0][1][0] - Ex[0][0][0] + Ey[1][0][0] - Ey[0][0][0] + Ez[0][0][1] - Ez[0][0][0])
    
#pragma overtile end

  double Stop = rtclock();

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";
  
  delete [] Ex;
  delete [] Ey;
  delete [] Ez;
  delete [] Hz;

  return 0;
}
