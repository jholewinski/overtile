#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <sys/time.h>
#include <iostream>

#ifndef PROBLEM_SIZE
#define PROBLEM_SIZE 4000
#endif

#ifndef TIME_STEPS
#define TIME_STEPS 50
#endif


inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}


template <typename T>
bool CompareResult(T *Result, T *Reference, size_t Size) {
  
  T ErrorNorm = 0.0;
  T RefNorm   = 0.0;

  for (unsigned i = 0; i < Size; ++i) {
    float Diff  = Result[i] - Reference[i];
    ErrorNorm  += Diff*Diff;
    RefNorm    += Reference[i]*Reference[i];
  }

  ErrorNorm = std::sqrt(ErrorNorm);
  RefNorm   = std::sqrt(RefNorm);

  std::cout << "Error Norm:  " << ErrorNorm << "\n";
  std::cout << "Ref Norm:    " << RefNorm << "\n";
 
  if(std::abs(RefNorm) < 1e-7) {
    std::cout << "FAIL!\n";
    return false;
  }
  else if((ErrorNorm / RefNorm) > 1e-2) {
    std::cout << "FAIL!\n";
    return false;
  }
  else {
    std::cout << "OK\n";
    return true;
  }
}


void reference(float *u, const float *f, float *g) {

  int   m, n;
  int   Iter;
  float r;
  
  const int M = PROBLEM_SIZE;
  const int N = PROBLEM_SIZE;

  const float DT      = 5.0f;
  const float EPSILON = 1.0E-20f;

  float sigma  = 1.00001f;
  float sigma2 = sigma*sigma;
  float lambda = 1.00001f;
  float gamma  = lambda/sigma2;


  float *Temp = new float[M*N];

  memcpy(Temp, u, sizeof(float)*M*N);
  
#define SQR(x) ((x)*(x))
  
    for(Iter = 1; Iter <= TIME_STEPS; Iter++)
    {
        /* Macros for referring to pixel neighbors */
        #define CENTER   (m+n*M)
        #define RIGHT    (m+n*M+M)
        #define LEFT     (m+n*M-M)
        #define DOWN     (m+n*M+1)
        #define UP       (m+n*M-1)        
        
        /* Approximate g = 1/|grad u| */
        for(n = 1; n < N-1; n++)
            for(m = 1; m < M-1; m++)
                g[CENTER] = 1.0/sqrt( EPSILON
                   + SQR(u[CENTER] - u[RIGHT])
                   + SQR(u[CENTER] - u[LEFT])
                   + SQR(u[CENTER] - u[DOWN])
                   + SQR(u[CENTER] - u[UP]) );        
        
        
        for(n = 1; n < N-1; n++)
            for(m = 1; m < M-1; m++)
            {
                /* Evaluate r = I1(u*f/sigma^2) / I0(u*f/sigma^2) with
                 a cubic rational approximation. */
                r = u[CENTER]*f[CENTER]/sigma2;
                r = ( r*(2.38944 + r*(0.950037 + r)) )
                   / ( 4.65314 + r*(2.57541 + r*(1.48937 + r)) );
                /* Update u */           

                Temp[CENTER] = ( u[CENTER] + DT*(u[RIGHT]*g[RIGHT]
                   + u[LEFT]*g[LEFT] + u[DOWN]*g[DOWN] + u[UP]*g[UP] 
                   + gamma*f[CENTER]*r) ) /
                   (1.0 + DT*(g[RIGHT] + g[LEFT] + g[DOWN] + g[UP] + gamma));
                
            }


        memcpy(u, Temp, sizeof(float)*M*N);
        
    }


  delete [] Temp;
}


int main() {

  const int Dim_0 = PROBLEM_SIZE;
  const int Dim_1 = PROBLEM_SIZE;
  
  float *G = new float[Dim_0*Dim_1];
  float *U = new float[Dim_0*Dim_1];
  float *F = new float[Dim_0*Dim_1];

  srand(time(NULL));
  
  for (int i = 0; i < Dim_0*Dim_1; ++i) {
    G[i] = 0.0f;
    F[i] = (float)rand() / (float)(RAND_MAX+1.0f) * 10.0f;
    U[i] = F[i];
  }
  
  float *RefU = new float[Dim_0*Dim_1];
  float *RefG = new float[Dim_0*Dim_1];

  memcpy(RefU, U, sizeof(float)*Dim_0*Dim_1);
  memcpy(RefG, G, sizeof(float)*Dim_0*Dim_1);


  double RefStart = rtclock();

#ifdef REF_TEST
  reference(RefU, F, RefG);
#endif
  
  double RefStop = rtclock();

#ifndef REF_TEST  
  hipDeviceSynchronize();
#endif

  double Start = rtclock();

#ifndef REF_TEST
  
#pragma overtile begin time_steps:TIME_STEPS block:32,8 tile:1,4 time:2

  program rician2d is
  grid 2
  field G float inout
  field U float inout
  field F float in

  F[0:0][0:0] = F[0][0]
    
  G[1:1][1:1] = 
    
    let left   = ((U[0][0] - U[0][-1])*(U[0][0] - U[0][-1])) in
    let right  = ((U[0][0] - U[0][1])*(U[0][0] - U[0][1])) in
    let top    = ((U[0][0] - U[-1][0])*(U[0][0] - U[-1][0])) in
    let bottom = ((U[0][0] - U[1][0])*(U[0][0] - U[1][0])) in
    let epsilon = 1.0e-20 in

      rsqrtf(epsilon + right + left + top + bottom)

  U[1:1][1:1] =

    let DT      = 5.0 in
    let sigma   = 1.00001 in
    let sigma2  = sigma*sigma in
    let lambda  = 1.00001 in
    let gamma   = lambda/sigma2 in
    let r_inner = U[0][0]*F[0][0]/sigma2 in
    let r       = (r_inner*(2.38944 + r_inner*(0.950037+r_inner))) / (4.65314 + r_inner*(2.57541 + r_inner*(1.48937 + r_inner))) in

    let left = U[-1][0]*G[-1][0] in
    let right  = U[1][0]*G[1][0] in
    let top    = U[0][-1]*G[0][-1] in
    let bottom = U[0][1]*G[0][1] in

    (U[0][0] + DT*(right + left + top + bottom + gamma*F[0][0]*r)) /
      (1.0 + DT*(G[0][1] + G[0][-1] + G[-1][0] + G[1][0] + gamma))

    
#pragma overtile end

#endif

  double Stop = rtclock();

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";
  std::cout << "Ref Elapsed: " << (RefStop - RefStart) << "\n";


  std::cout << "Check U...\n";
  CompareResult(U, RefU, Dim_0*Dim_1);


#ifdef REF_TEST
  double GStencils = (Dim_0-2)*(Dim_1-2)*(double)TIME_STEPS/1e9/(RefStop-RefStart);
  std::cout << "GStencils/sec: " << GStencils << "\n";
#else
  double GStencils = (Dim_0-2)*(Dim_1-2)*(double)TIME_STEPS/1e9/(Stop-Start);
  std::cout << "GStencils/sec: " << GStencils << "\n";
#endif


  
  delete [] G;
  delete [] U;
  delete [] F;
  delete [] RefG;
  delete [] RefU;

  return 0;
}
