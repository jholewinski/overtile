
#include <sys/time.h>
#include <iostream>

#ifndef PROBLEM_SIZE
#define PROBLEM_SIZE 300
#endif

#ifndef TIME_STEPS
#define TIME_STEPS 100
#endif


inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}





int main() {

  const int Dim_0 = PROBLEM_SIZE;
  const int Dim_1 = PROBLEM_SIZE;
  const int Dim_2 = PROBLEM_SIZE;
  
  float *A = new float[Dim_0*Dim_1*Dim_2];

  hipDeviceSynchronize();

  double Start = rtclock();
  
#pragma overtile begin time_steps:TIME_STEPS block:16,4,4 tile:1,8,1 time:1

  program j3d is
  grid 3
  field A float inout
    
    A[1:1][1:1][1:1] = 0.143 * (A[-1][0][0] + A[0][0][0] + A[1][0][0] + A[0][-1][0] + A[0][1][0] + A[0][0][-1] + A[0][0][1])

#pragma overtile end

  double Stop = rtclock();

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";
  
  delete [] A;

  return 0;
}
