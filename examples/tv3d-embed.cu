#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <sys/time.h>
#include <iostream>

#ifndef PROBLEM_SIZE
#define PROBLEM_SIZE 200
#endif

#ifndef TIME_STEPS
#define TIME_STEPS 50
#endif


inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}


template <typename T>
bool CompareResult(T *Result, T *Reference, size_t Size) {
  
  T ErrorNorm = 0.0;
  T RefNorm   = 0.0;

  for (unsigned i = 0; i < Size; ++i) {
    float Diff  = Result[i] - Reference[i];
    ErrorNorm  += Diff*Diff;
    RefNorm    += Reference[i]*Reference[i];
  }

  ErrorNorm = std::sqrt(ErrorNorm);
  RefNorm   = std::sqrt(RefNorm);

  std::cout << "Error Norm:  " << ErrorNorm << "\n";
  std::cout << "Ref Norm:    " << RefNorm << "\n";
 
  if(std::abs(RefNorm) < 1e-7) {
    std::cout << "FAIL!\n";
    return false;
  }
  else if((ErrorNorm / RefNorm) > 1e-2) {
    std::cout << "FAIL!\n";
    return false;
  }
  else {
    std::cout << "OK\n";
    return true;
  }
}


void reference(float *image, const float *sumA) {

  int   p, n, m;
  int   Iter;
  float r;
  
  const int M = PROBLEM_SIZE;
  const int N = PROBLEM_SIZE;
  const int P = PROBLEM_SIZE;

  const float DT      = 5.0f;
  const float EPSILON = 1.0E-20f;

  float sigma  = 1.00001f;
  float sigma2 = sigma*sigma;
  float lambda = 1.00001f;
  float gamma  = lambda/sigma2;

  const float alpha = 1.001f;


  float *Temp = new float[M*N*P];

  memcpy(Temp, image, sizeof(float)*M*N*P);
  
#define SQR(x) ((x)*(x))
  
  int N_x = PROBLEM_SIZE;
  int N_y = PROBLEM_SIZE;
  int N_z = PROBLEM_SIZE;
  
  float c1, c2, c3, c4;
  float eps = 1e-8;

#define REF(i,j,k) ((i)*PROBLEM_SIZE*PROBLEM_SIZE+(j)*PROBLEM_SIZE+(k))
  
  //      DoubleArray image_old()
  for (int i   = 0; i < TIME_STEPS; i ++)
  {
    for (int j = 1; j < N_x-1; j ++)
      for (int k = 1; k < N_y-1; k ++)
        for (int l = 1; l < N_z-1; l ++)
        {
            c1           = image[REF(j,k,l)]/sqrtf(eps + SQR(image[REF(j + 1,k,l)] - image[REF(j,k,l)]) + SQR(image[REF(j,k + 1,l)] - image[REF(j,k,l)]) + SQR(image[REF(j,k,l + 1)] - image[REF(j,k,l)]))/sumA[REF(j,k,l)];
            c2           = image[REF(j,k,l)]/sqrtf(eps + SQR(image[REF(j,k,l)] - image[REF(j - 1,k,l)]) + SQR(image[REF(j - 1,k + 1,l)] - image[REF(j - 1,k,l)]) + SQR(image[REF(j - 1,k,l + 1)] - image[REF(j - 1,k,l)]))/sumA[REF(j,k,l)];
            c3           = image[REF(j,k,l)]/sqrtf(eps + SQR(image[REF(j + 1,k - 1,l)] - image[REF(j,k - 1,l)]) + SQR(image[REF(j,k,l)] - image[REF(j,k - 1,l)]) + SQR(image[REF(j,k - 1,l + 1)] - image[REF(j,k - 1,l)]))/sumA[REF(j,k,l)];
            c4           = image[REF(j,k,l)]/sqrtf(eps + SQR(image[REF(j + 1,k,l - 1)] - image[REF(j,k,l - 1)]) + SQR(image[REF(j,k + 1,l - 1)] - image[REF(j,k,l - 1)]) + SQR(image[REF(j,k,l)] - image[REF(j,k,l - 1)]))/sumA[REF(j,k,l)];
            Temp[REF(j,k,l)] = (alpha*image[REF(j,k,l)] + c1*(image[REF(j + 1,k,l) ]+ image[REF(j,k + 1,l)] + image[REF(j,k,l + 1)]) + c2*image[REF(j - 1,k,l)] + c3*image[REF(j,k - 1,l)] + c4*image[REF(j,k,l - 1)])/(alpha + 3.0f*c1 + c2 + c3 + c4);
        }

    memcpy(image, Temp, sizeof(float)*N_x*N_y*N_z);
  }
  


  delete [] Temp;
}


int main() {

  const int Dim_0 = PROBLEM_SIZE;
  const int Dim_1 = PROBLEM_SIZE;
  const int Dim_2 = PROBLEM_SIZE;
  
  float *Image = new float[Dim_0*Dim_1*Dim_2];
  float *SumA = new float[Dim_0*Dim_1*Dim_2];

  srand(time(NULL));
  
  for (int i = 0; i < Dim_0*Dim_1*Dim_2; ++i) {
    Image[i] = (float)rand() / (float)(RAND_MAX+1.0f) * 10.0f;
    SumA[i] = (float)rand() / (float)(RAND_MAX+1.0f) * 10.0f;
  }
  
  float *RefImage = new float[Dim_0*Dim_1*Dim_2];
  float *RefSumA = new float[Dim_0*Dim_1*Dim_2];

  memcpy(RefImage, Image, sizeof(float)*Dim_0*Dim_1*Dim_2);
  memcpy(RefSumA, SumA, sizeof(float)*Dim_0*Dim_1*Dim_2);


  double RefStart = rtclock();

#ifdef REF_TEST
  reference(RefImage, RefSumA);
#endif

  double RefStop = rtclock();


#ifndef REF_TEST
  hipDeviceSynchronize();
#endif

  double Start = rtclock();

#ifndef REF_TEST
  
#pragma overtile begin time_steps:TIME_STEPS block:8,8,8 tile:1,1,1 time:1

  program tv3d is
    
  grid 3

  field Image float inout
  field SumA float in

    Image[1:1][1:1] =
    let eps = 1e-8 in
    let alpha = 1.001 in

    let c1 = Image[0][0][0] / sqrtf(eps + SQR(Image[1][0][0] - Image[0][0][0]) + SQR(Image[0][1][0] - Image[0][0][0]) + SQR(Image[0][0][1] - Image[0][0][0])) / SumA[0][0][0] in
    let c2 = Image[0][0][0] / sqrtf(eps + SQR(Image[0][0][0] - Image[-1][0][0]) + SQR(Image[-1][1][0] - Image[-1][0][0]) + SQR(Image[-1][0][1] - Image[-1][0][0])) / SumA[0][0][0] in
    let c3 = Image[0][0][0] / sqrtf(eps + SQR(Image[1][-1][0] - Image[0][-1][0]) + SQR(Image[0][0][0] - Image[0][-1][0]) + SQR(Image[0][-1][1] - Image[0][-1][0])) / SumA[0][0][0] in
    let c4 = Image[0][0][0] / sqrtf(eps + SQR(Image[1][0][-1] - Image[0][0][-1]) + SQR(Image[0][1][-1] - Image[0][0][-1]) + SQR(Image[0][0][0] - Image[0][0][-1])) / SumA[0][0][0] in

    (alpha*Image[0][0][0] + c1*(Image[1][0][0] + Image[0][1][0] + Image[0][0][1]) + c2*Image[-1][0][0] + c3*Image[0][-1][0] + c4*Image[0][0][-1])/(alpha + 3.0*c1 + c2 + c3 + c4);

    
#pragma overtile end

#endif

  double Stop = rtclock();

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";
  std::cout << "Ref Elapsed: " << (RefStop - RefStart) << "\n";


  //std::cout << "Check U...\n";
  //CompareResult(U, RefU, Dim_0*Dim_1*Dim_2);

#ifdef REF_TEST
  double GStencils = (Dim_0-2)*(Dim_1-2)*(Dim_2-2)*(double)TIME_STEPS/1e9/(RefStop-RefStart);
  std::cout << "GStencils/sec: " << GStencils << "\n";
#else
  double GStencils = (Dim_0-2)*(Dim_1-2)*(Dim_2-2)*(double)TIME_STEPS/1e9/(Stop-Start);
  std::cout << "GStencils/sec: " << GStencils << "\n";
#endif

  
  delete [] Image;
  delete [] SumA;
  delete [] RefSumA;
  delete [] RefImage;

  return 0;
}
