#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <sys/time.h>
#include <iostream>

#ifndef PROBLEM_SIZE
#define PROBLEM_SIZE 4000
#endif

#ifndef TIME_STEPS
#define TIME_STEPS 50
#endif


inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}


template <typename T>
bool CompareResult(T *Result, T *Reference, size_t Size) {
  
  T ErrorNorm = 0.0;
  T RefNorm   = 0.0;

  for (unsigned i = 0; i < Size; ++i) {
    double Diff  = Result[i] - Reference[i];
    ErrorNorm  += Diff*Diff;
    RefNorm    += Reference[i]*Reference[i];
  }

  ErrorNorm = std::sqrt(ErrorNorm);
  RefNorm   = std::sqrt(RefNorm);

  std::cout << "Error Norm:  " << ErrorNorm << "\n";
  std::cout << "Ref Norm:    " << RefNorm << "\n";
 
  if(std::abs(RefNorm) < 1e-7) {
    std::cout << "FAIL!\n";
    return false;
  }
  else if((ErrorNorm / RefNorm) > 1e-2) {
    std::cout << "FAIL!\n";
    return false;
  }
  else {
    std::cout << "OK\n";
    return true;
  }
}


void reference(double *u, const double *f, const double *V) {

  int   m, n;
  int   Iter;
  double r;
  
  const int M = PROBLEM_SIZE;
  const int N = PROBLEM_SIZE;

  double *Temp = new double[N*M];

  const double epi2  = 1e-8;
  const double alpha = 1.001;
  
#define REF(i,j) ((i)*M+(j))
#define SQR(x)   ((x)*(x))
  
  for (int t = 0; t < TIME_STEPS; ++t) {
    for (int i = 1; i < N-1; ++i) {
      for (int j = 1; j < M-1; ++j) {

        double c1 = u[REF(i,j)] / sqrt(epi2 + SQR(u[REF(i+1,j)]-u[REF(i,j)]) + SQR(u[REF(i,j+1)]-u[REF(i,j)])) / V[REF(i,j)];
        double c2 = u[REF(i,j)] / sqrt(epi2 + SQR(u[REF(i,j)]-u[REF(i-1,j)]) + SQR(u[REF(i-1,j+1)]-u[REF(i-1,j)])) / V[REF(i,j)];
        double c3 = u[REF(i,j)] / sqrt(epi2 + SQR(u[REF(i+1,j)]-u[REF(i,j)]) + SQR(u[REF(i,j+1)]-u[REF(i,j)])) / V[REF(i,j)];
        double c4 = u[REF(i,j)] / sqrt(epi2 + SQR(u[REF(i+1,j-1)]-u[REF(i,j-1)]) + SQR(u[REF(i,j)]-u[REF(i,j-1)])) / V[REF(i,j)];

        
        Temp[REF(i,j)] = 1.0 / (alpha+c1+c2+c3+c4)*(alpha*f[REF(i,j)]+c1*u[REF(i+1,j)]+c2*u[REF(i-1,j)]+c3*u[REF(i,j+1)]+c4*u[REF(i,j-1)]);
      }
    }

    memcpy(u, Temp, sizeof(double)*N*M);
  }

  delete [] Temp;
}


int main() {

  const int Dim_0 = PROBLEM_SIZE;
  const int Dim_1 = PROBLEM_SIZE;
  
  double *V = new double[Dim_0*Dim_1];
  double *u = new double[Dim_0*Dim_1];
  double *f = new double[Dim_0*Dim_1];

  srand(time(NULL));
  
  for (int i = 0; i < Dim_0*Dim_1; ++i) {
    V[i] = (double)rand() / (double)(RAND_MAX+1.0f) * 10.0f;
    f[i] = (double)rand() / (double)(RAND_MAX+1.0f) * 10.0f;
    u[i] = (double)rand() / (double)(RAND_MAX+1.0f) * 10.0f;
  }
  
  double *RefU = new double[Dim_0*Dim_1];
  double *RefF = new double[Dim_0*Dim_1];
  double *RefV = new double[Dim_0*Dim_1];

  memcpy(RefU, u, sizeof(double)*Dim_0*Dim_1);
  memcpy(RefF, f, sizeof(double)*Dim_0*Dim_1);
  memcpy(RefV, V, sizeof(double)*Dim_0*Dim_1);


  double RefStart = rtclock();

#ifdef REF_TEST
  reference(RefU, RefF, RefV);
#endif
  
  double RefStop = rtclock();

#ifndef REF_TEST  
  hipDeviceSynchronize();
#endif

  double Start = rtclock();

#ifndef REF_TEST
  
#pragma overtile begin time_steps:TIME_STEPS block:32,8 tile:1,6 time:1

  
program tv2d is

  grid 2

  field u double inout
  field V double in
  field f double in


  u[1:1][1:1] = 
  let epi2    = 1e-8 in
  let alpha   = 1.001 in
  let c1      = u[0][0] / sqrt(epi2 + SQR(u[1][0]-u[0][0]) + SQR(u[0][1]-u[0][0])) / V[0][0] in
  let c2      = u[0][0] / sqrt(epi2 + SQR(u[0][0]-u[-1][0]) + SQR(u[-1][1]-u[-1][0])) / V[0][0] in
  let c3      = u[0][0] / sqrt(epi2 + SQR(u[1][0]-u[0][0]) + SQR(u[0][1]-u[0][0])) / V[0][0] in
  let c4      = u[0][0] / sqrt(epi2 + SQR(u[1][-1]-u[0][-1]) + SQR(u[0][0]-u[0][-1])) / V[0][0] in
       
    1.0 / (alpha+c1+c2+c3+c4)*(alpha*f[0][0]+c1*u[1][0]+c2*u[-1][0]+c3*u[0][1]+c4*u[0][-1])
    
#pragma overtile end

#endif

  double Stop = rtclock();

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";
  std::cout << "Ref Elapsed: " << (RefStop - RefStart) << "\n";


  std::cout << "Check U...\n";
  CompareResult(u, RefU, Dim_0*Dim_1);


#ifdef REF_TEST
  double GStencils = (Dim_0-2)*(Dim_1-2)*(double)TIME_STEPS/1e9/(RefStop-RefStart);
  std::cout << "GStencils/sec: " << GStencils << "\n";
#else
  double GStencils = (Dim_0-2)*(Dim_1-2)*(double)TIME_STEPS/1e9/(Stop-Start);
  std::cout << "GStencils/sec: " << GStencils << "\n";
#endif


  
  delete [] V;
  delete [] u;
  delete [] f;
  delete [] RefF;
  delete [] RefU;
  delete [] RefV;

  return 0;
}
