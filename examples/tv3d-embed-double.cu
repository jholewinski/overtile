#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <sys/time.h>
#include <iostream>

#ifndef PROBLEM_SIZE
#define PROBLEM_SIZE 200
#endif

#ifndef TIME_STEPS
#define TIME_STEPS 50
#endif


inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}


template <typename T>
bool CompareResult(T *Result, T *Reference, size_t Size) {
  
  T ErrorNorm = 0.0;
  T RefNorm   = 0.0;

  for (unsigned i = 0; i < Size; ++i) {
    double Diff  = Result[i] - Reference[i];
    ErrorNorm  += Diff*Diff;
    RefNorm    += Reference[i]*Reference[i];
  }

  ErrorNorm = std::sqrt(ErrorNorm);
  RefNorm   = std::sqrt(RefNorm);

  std::cout << "Error Norm:  " << ErrorNorm << "\n";
  std::cout << "Ref Norm:    " << RefNorm << "\n";
 
  if(std::abs(RefNorm) < 1e-7) {
    std::cout << "FAIL!\n";
    return false;
  }
  else if((ErrorNorm / RefNorm) > 1e-2) {
    std::cout << "FAIL!\n";
    return false;
  }
  else {
    std::cout << "OK\n";
    return true;
  }
}


void reference(double *image, const double *sumA) {

  int   p, n, m;
  int   Iter;
  double r;
  
  const int M = PROBLEM_SIZE;
  const int N = PROBLEM_SIZE;
  const int P = PROBLEM_SIZE;

  const double DT      = 5.0f;
  const double EPSILON = 1.0E-20f;

  double sigma  = 1.00001f;
  double sigma2 = sigma*sigma;
  double lambda = 1.00001f;
  double gamma  = lambda/sigma2;

  const double alpha = 1.001f;


  double *Temp = new double[M*N*P];

  memcpy(Temp, image, sizeof(double)*M*N*P);
  
#define SQR(x) ((x)*(x))
  
  int N_x = PROBLEM_SIZE;
  int N_y = PROBLEM_SIZE;
  int N_z = PROBLEM_SIZE;
  
  double c1, c2, c3, c4;
  double eps = 1e-8;

#define REF(i,j,k) ((i)*PROBLEM_SIZE*PROBLEM_SIZE+(j)*PROBLEM_SIZE+(k))
  
  //      DoubleArray image_old()
  for (int i   = 0; i < TIME_STEPS; i ++)
  {
    for (int j = 1; j < N_x-1; j ++)
      for (int k = 1; k < N_y-1; k ++)
        for (int l = 1; l < N_z-1; l ++)
        {
            c1           = image[REF(j,k,l)]/sqrt(eps + SQR(image[REF(j + 1,k,l)] - image[REF(j,k,l)]) + SQR(image[REF(j,k + 1,l)] - image[REF(j,k,l)]) + SQR(image[REF(j,k,l + 1)] - image[REF(j,k,l)]))/sumA[REF(j,k,l)];
            c2           = image[REF(j,k,l)]/sqrt(eps + SQR(image[REF(j,k,l)] - image[REF(j - 1,k,l)]) + SQR(image[REF(j - 1,k + 1,l)] - image[REF(j - 1,k,l)]) + SQR(image[REF(j - 1,k,l + 1)] - image[REF(j - 1,k,l)]))/sumA[REF(j,k,l)];
            c3           = image[REF(j,k,l)]/sqrt(eps + SQR(image[REF(j + 1,k - 1,l)] - image[REF(j,k - 1,l)]) + SQR(image[REF(j,k,l)] - image[REF(j,k - 1,l)]) + SQR(image[REF(j,k - 1,l + 1)] - image[REF(j,k - 1,l)]))/sumA[REF(j,k,l)];
            c4           = image[REF(j,k,l)]/sqrt(eps + SQR(image[REF(j + 1,k,l - 1)] - image[REF(j,k,l - 1)]) + SQR(image[REF(j,k + 1,l - 1)] - image[REF(j,k,l - 1)]) + SQR(image[REF(j,k,l)] - image[REF(j,k,l - 1)]))/sumA[REF(j,k,l)];
            Temp[REF(j,k,l)] = (alpha*image[REF(j,k,l)] + c1*(image[REF(j + 1,k,l) ]+ image[REF(j,k + 1,l)] + image[REF(j,k,l + 1)]) + c2*image[REF(j - 1,k,l)] + c3*image[REF(j,k - 1,l)] + c4*image[REF(j,k,l - 1)])/(alpha + 3.0*c1 + c2 + c3 + c4);
        }

    memcpy(image, Temp, sizeof(double)*N_x*N_y*N_z);
  }
  


  delete [] Temp;
}


int main() {

  const int Dim_0 = PROBLEM_SIZE;
  const int Dim_1 = PROBLEM_SIZE;
  const int Dim_2 = PROBLEM_SIZE;
  
  double *Image = new double[Dim_0*Dim_1*Dim_2];
  double *SumA = new double[Dim_0*Dim_1*Dim_2];

  srand(time(NULL));
  
  for (int i = 0; i < Dim_0*Dim_1*Dim_2; ++i) {
    Image[i] = (double)rand() / (double)(RAND_MAX+1.0f) * 10.0f;
    SumA[i] = (double)rand() / (double)(RAND_MAX+1.0f) * 10.0f;
  }
  
  double *RefImage = new double[Dim_0*Dim_1*Dim_2];
  double *RefSumA = new double[Dim_0*Dim_1*Dim_2];

  memcpy(RefImage, Image, sizeof(double)*Dim_0*Dim_1*Dim_2);
  memcpy(RefSumA, SumA, sizeof(double)*Dim_0*Dim_1*Dim_2);


  double RefStart = rtclock();

#ifdef REF_TEST
  reference(RefImage, RefSumA);
#endif

  double RefStop = rtclock();


#ifndef REF_TEST
  hipDeviceSynchronize();
#endif

  double Start = rtclock();

#ifndef REF_TEST
  
#pragma overtile begin time_steps:TIME_STEPS block:8,8,8 tile:1,1,1 time:1

  
  program tv3d is
    
  grid 3

  field Image double inout
  field SumA double in

    Image[1:1][1:1] =
    let eps = 1e-8 in
    let alpha = 1.001 in

    let c1 = Image[0][0][0] / sqrt(eps + SQR(Image[1][0][0] - Image[0][0][0]) + SQR(Image[0][1][0] - Image[0][0][0]) + SQR(Image[0][0][1] - Image[0][0][0])) / SumA[0][0][0] in
    let c2 = Image[0][0][0] / sqrt(eps + SQR(Image[0][0][0] - Image[-1][0][0]) + SQR(Image[-1][1][0] - Image[-1][0][0]) + SQR(Image[-1][0][1] - Image[-1][0][0])) / SumA[0][0][0] in
    let c3 = Image[0][0][0] / sqrt(eps + SQR(Image[1][-1][0] - Image[0][-1][0]) + SQR(Image[0][0][0] - Image[0][-1][0]) + SQR(Image[0][-1][1] - Image[0][-1][0])) / SumA[0][0][0] in
    let c4 = Image[0][0][0] / sqrt(eps + SQR(Image[1][0][-1] - Image[0][0][-1]) + SQR(Image[0][1][-1] - Image[0][0][-1]) + SQR(Image[0][0][0] - Image[0][0][-1])) / SumA[0][0][0] in

    (alpha*Image[0][0][0] + c1*(Image[1][0][0] + Image[0][1][0] + Image[0][0][1]) + c2*Image[-1][0][0] + c3*Image[0][-1][0] + c4*Image[0][0][-1])/(alpha + 3.0*c1 + c2 + c3 + c4);
    
#pragma overtile end

#endif

  double Stop = rtclock();

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";
  std::cout << "Ref Elapsed: " << (RefStop - RefStart) << "\n";


  //std::cout << "Check U...\n";
  //CompareResult(U, RefU, Dim_0*Dim_1*Dim_2);

#ifdef REF_TEST
  double GStencils = (Dim_0-2)*(Dim_1-2)*(Dim_2-2)*(double)TIME_STEPS/1e9/(RefStop-RefStart);
  std::cout << "GStencils/sec: " << GStencils << "\n";
#else
  double GStencils = (Dim_0-2)*(Dim_1-2)*(Dim_2-2)*(double)TIME_STEPS/1e9/(Stop-Start);
  std::cout << "GStencils/sec: " << GStencils << "\n";
#endif

  
  delete [] Image;
  delete [] SumA;
  delete [] RefSumA;
  delete [] RefImage;

  return 0;
}
