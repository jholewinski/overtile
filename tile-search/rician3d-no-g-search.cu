#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <sys/time.h>
#include <iostream>

#ifndef PROBLEM_SIZE
#define PROBLEM_SIZE 300
#endif

#ifndef TIME_STEPS
#define TIME_STEPS 100
#endif


inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

  
#define SQR(x) ((x)*(x))


int main() {

  const int Dim_0 = PROBLEM_SIZE;
  const int Dim_1 = PROBLEM_SIZE;
  const int Dim_2 = PROBLEM_SIZE;
  
  float *U = new float[Dim_0*Dim_1*Dim_2];
  float *F = new float[Dim_0*Dim_1*Dim_2];


  hipDeviceSynchronize();


  double Start = rtclock();

#pragma overtile begin time_steps:TIME_STEPS TILE_SIZE_PARAMS

  program rician3d is
  grid 3
  field U float inout
  field F float in

  U[1:1][1:1][1:1] =

    let DT      = 5.0 in
    let sigma   = 1.00001 in
    let sigma2  = sigma*sigma in
    let lambda  = 1.00001 in
    let gamma   = lambda/sigma2 in
    let r_inner = U[0][0][0]*F[0][0][0]/sigma2 in
    let r       = (r_inner*(2.38944 + r_inner*(0.950037+r_inner))) / (4.65314 + r_inner*(2.57541 + r_inner*(1.48937 + r_inner))) in

    let epsilon = 1.0e-20 in


    let left_1_0_0   = ((U[1][0][0] - U[1][0][-1])*(U[1][0][0] - U[1][0][-1])) in
    let right_1_0_0  = ((U[1][0][0] - U[1][0][1])*(U[1][0][0] - U[1][0][1])) in
    let top_1_0_0    = ((U[1][0][0] - U[1][-1][0])*(U[1][0][0] - U[1][-1][0])) in
    let bottom_1_0_0 = ((U[1][0][0] - U[1][1][0])*(U[1][0][0] - U[1][1][0])) in
    let back_1_0_0   = ((U[1][0][0] - U[0][0][0])*(U[1][0][0] - U[0][0][0])) in
    let front_1_0_0  = ((U[1][0][0] - U[2][0][0])*(U[1][0][0] - U[2][0][0])) in
    let g_1_0_0      = rsqrtf(epsilon + right_1_0_0 + left_1_0_0 + top_1_0_0 + bottom_1_0_0 + back_1_0_0 + front_1_0_0) in

    let left_m1_0_0   = ((U[-1][0][0] - U[-1][0][-1])*(U[-1][0][0] - U[-1][0][-1])) in
    let right_m1_0_0  = ((U[-1][0][0] - U[-1][0][1])*(U[-1][0][0] - U[-1][0][1])) in
    let top_m1_0_0    = ((U[-1][0][0] - U[-1][-1][0])*(U[-1][0][0] - U[-1][-1][0])) in
    let bottom_m1_0_0 = ((U[-1][0][0] - U[-1][1][0])*(U[-1][0][0] - U[-1][1][0])) in
    let back_m1_0_0   = ((U[-1][0][0] - U[-2][0][0])*(U[-1][0][0] - U[-2][0][0])) in
    let front_m1_0_0  = ((U[-1][0][0] - U[0][0][0])*(U[-1][0][0] - U[0][0][0])) in
    let g_m1_0_0      = rsqrtf(epsilon + right_m1_0_0 + left_m1_0_0 + top_m1_0_0 + bottom_m1_0_0 + back_m1_0_0 + front_m1_0_0) in

    let left_0_1_0   = ((U[0][1][0] - U[0][1][-1])*(U[0][1][0] - U[0][1][-1])) in
    let right_0_1_0  = ((U[0][1][0] - U[0][1][1])*(U[0][1][0] - U[0][1][1])) in
    let top_0_1_0    = ((U[0][1][0] - U[0][0][0])*(U[0][1][0] - U[0][0][0])) in
    let bottom_0_1_0 = ((U[0][1][0] - U[0][2][0])*(U[0][1][0] - U[0][2][0])) in
    let back_0_1_0   = ((U[0][1][0] - U[-1][1][0])*(U[0][1][0] - U[-1][1][0])) in
    let front_0_1_0  = ((U[0][1][0] - U[1][1][0])*(U[0][1][0] - U[1][1][0])) in
    let g_0_1_0      = rsqrtf(epsilon + right_0_1_0 + left_0_1_0 + top_0_1_0 + bottom_0_1_0 + back_0_1_0 + front_0_1_0) in

    let left_0_m1_0   = ((U[0][-1][0] - U[0][-1][-1])*(U[0][-1][0] - U[0][-1][-1])) in
    let right_0_m1_0  = ((U[0][-1][0] - U[0][-1][1])*(U[0][-1][0] - U[0][-1][1])) in
    let top_0_m1_0    = ((U[0][-1][0] - U[0][-2][0])*(U[0][-1][0] - U[0][-2][0])) in
    let bottom_0_m1_0 = ((U[0][-1][0] - U[0][0][0])*(U[0][-1][0] - U[0][0][0])) in
    let back_0_m1_0   = ((U[0][-1][0] - U[-1][-1][0])*(U[0][-1][0] - U[-1][-1][0])) in
    let front_0_m1_0  = ((U[0][-1][0] - U[1][-1][0])*(U[0][-1][0] - U[1][-1][0])) in
    let g_0_m1_0      = rsqrtf(epsilon + right_0_m1_0 + left_0_m1_0 + top_0_m1_0 + bottom_0_m1_0 + back_0_m1_0 + front_0_m1_0) in

    let left_0_0_1   = ((U[0][0][1] - U[0][0][0])*(U[0][0][1] - U[0][0][0])) in
    let right_0_0_1  = ((U[0][0][1] - U[0][0][2])*(U[0][0][1] - U[0][0][2])) in
    let top_0_0_1    = ((U[0][0][1] - U[0][-1][1])*(U[0][0][1] - U[0][-1][1])) in
    let bottom_0_0_1 = ((U[0][0][1] - U[0][1][1])*(U[0][0][1] - U[0][1][1])) in
    let back_0_0_1   = ((U[0][0][1] - U[-1][0][1])*(U[0][0][1] - U[-1][0][1])) in
    let front_0_0_1  = ((U[0][0][1] - U[1][0][1])*(U[0][0][1] - U[1][0][1])) in
    let g_0_0_1      = rsqrtf(epsilon + right_0_0_1 + left_0_0_1 + top_0_0_1 + bottom_0_0_1 + back_0_0_1 + front_0_0_1) in

    let left_0_0_m1   = ((U[0][0][-1] - U[0][0][-2])*(U[0][0][-1] - U[0][0][-2])) in
    let right_0_0_m1  = ((U[0][0][-1] - U[0][0][0])*(U[0][0][-1] - U[0][0][0])) in
    let top_0_0_m1    = ((U[0][0][-1] - U[0][-1][-1])*(U[0][0][-1] - U[0][-1][1])) in
    let bottom_0_0_m1 = ((U[0][0][-1] - U[0][1][-1])*(U[0][0][-1] - U[0][1][1])) in
    let back_0_0_m1   = ((U[0][0][-1] - U[-1][0][-1])*(U[0][0][-1] - U[-1][0][1])) in
    let front_0_0_m1  = ((U[0][0][-1] - U[1][0][-1])*(U[0][0][-1] - U[1][0][1])) in
    let g_0_0_m1      = rsqrtf(epsilon + right_0_0_m1 + left_0_0_m1 + top_0_0_m1 + bottom_0_0_m1 + back_0_0_m1 + front_0_0_m1) in



    
    let left   = U[0][-1][0]*g_0_m1_0 in
    let right  = U[0][1][0]*g_0_1_0 in
    let top    = U[0][0][-1]*g_0_0_m1 in
    let bottom = U[0][0][1]*g_0_0_1 in
    let back   = U[-1][0][0]*g_m1_0_0 in
    let front  = U[1][0][0]*g_1_0_0 in

    (U[0][0][0] + DT*(right + left + top + bottom + back + front + gamma*F[0][0][0]*r)) /
      (1.0 + DT*(g_0_0_1 + g_0_0_m1 + g_0_m1_0 + g_0_1_0 + g_m1_0_0 + g_1_0_0 + gamma))

    
#pragma overtile end


  double Stop = rtclock();

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";


  double GStencils = (Dim_0-2)*(Dim_1-2)*(Dim_2-2)*(double)TIME_STEPS/1e9/(Stop-Start);
  std::cout << "GStencils/sec: " << GStencils << "\n";

  
  delete [] U;
  delete [] F;

  return 0;
}
