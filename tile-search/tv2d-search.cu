#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <sys/time.h>
#include <iostream>

#ifndef PROBLEM_SIZE
#define PROBLEM_SIZE 6000
#endif

#ifndef TIME_STEPS
#define TIME_STEPS 100
#endif


inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

#define SQR(x) ((x)*(x))

int main() {

  const int Dim_0 = PROBLEM_SIZE;
  const int Dim_1 = PROBLEM_SIZE;
  
  float *V = new float[Dim_0*Dim_1];
  float *u = new float[Dim_0*Dim_1];
  float *f = new float[Dim_0*Dim_1];

  hipDeviceSynchronize();

  double Start = rtclock();

#pragma overtile begin time_steps:TIME_STEPS block:32,8 tile:1,6 time:1

program tv2d is

  grid 2

  field u float inout
  field V float in
  field f float in


  u[1:1][1:1] = 
  let epi2    = 1e-8 in
  let alpha   = 1.001 in
  let c1      = u[0][0] / sqrtf(epi2 + SQR(u[1][0]-u[0][0]) + SQR(u[0][1]-u[0][0])) / V[0][0] in
  let c2      = u[0][0] / sqrtf(epi2 + SQR(u[0][0]-u[-1][0]) + SQR(u[-1][1]-u[-1][0])) / V[0][0] in
  let c3      = u[0][0] / sqrtf(epi2 + SQR(u[1][0]-u[0][0]) + SQR(u[0][1]-u[0][0])) / V[0][0] in
  let c4      = u[0][0] / sqrtf(epi2 + SQR(u[1][-1]-u[0][-1]) + SQR(u[0][0]-u[0][-1])) / V[0][0] in
       
    1.0 / (alpha+c1+c2+c3+c4)*(alpha*f[0][0]+c1*u[1][0]+c2*u[-1][0]+c3*u[0][1]+c4*u[0][-1])

#pragma overtile end

  double Stop = rtclock();

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";

  double GStencils = (Dim_0-2)*(Dim_1-2)*(double)TIME_STEPS/1e9/(Stop-Start);
  std::cout << "GStencils/sec: " << GStencils << "\n";
  
  delete [] V;
  delete [] u;
  delete [] f;

  return 0;
}
