#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <sys/time.h>
#include <iostream>

#ifndef PROBLEM_SIZE
#define PROBLEM_SIZE 6000
#endif

#ifndef TIME_STEPS
#define TIME_STEPS 100
#endif


inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}


#define SQR(x) ((x)*(x))


int main() {

  const int Dim_0 = PROBLEM_SIZE;
  const int Dim_1 = PROBLEM_SIZE;
  
  double *U = new double[Dim_0*Dim_1];
  double *F = new double[Dim_0*Dim_1];




  hipDeviceSynchronize();

  double Start = rtclock();

#ifndef REF_TEST
  
#pragma overtile begin time_steps:TIME_STEPS TILE_SIZE_PARAMS

  program rician2d is
  grid 2
  field U double inout
  field F double in

  U[1:1][1:1] =

    let DT      = 5.0 in
    let sigma   = 1.00001 in
    let sigma2  = sigma*sigma in
    let lambda  = 1.00001 in
    let gamma   = lambda/sigma2 in
    let r_inner = U[0][0]*F[0][0]/sigma2 in
    let r       = (r_inner*(2.38944 + r_inner*(0.950037+r_inner))) / (4.65314 + r_inner*(2.57541 + r_inner*(1.48937 + r_inner))) in
    let epsilon = 1.0e-20 in


    let left_0_m1   = ((U[0][-1] - U[0][-2])*(U[0][-1] - U[0][-2])) in
    let right_0_m1  = ((U[0][-1] - U[0][0])*(U[0][-1] - U[0][0])) in
    let top_0_m1    = ((U[0][-1] - U[-1][-1])*(U[0][-1] - U[-1][-1])) in
    let bottom_0_m1 = ((U[0][-1] - U[1][-1])*(U[0][-1] - U[1][-1])) in
    let g_0_m1      = rsqrt(epsilon + right_0_m1 + left_0_m1 + top_0_m1 + bottom_0_m1) in
    
    let left_0_p1   = ((U[0][1] - U[0][0])*(U[0][1] - U[0][0])) in
    let right_0_p1  = ((U[0][1] - U[0][2])*(U[0][1] - U[0][2])) in
    let top_0_p1    = ((U[0][1] - U[-1][1])*(U[0][1] - U[-1][1])) in
    let bottom_0_p1 = ((U[0][1] - U[1][1])*(U[0][1] - U[1][1])) in
    let g_0_p1      = rsqrt(epsilon + right_0_p1 + left_0_p1 + top_0_p1 + bottom_0_p1) in

    let left_m1_0   = ((U[-1][0] - U[-1][-1])*(U[-1][0] - U[1][-1])) in
    let right_m1_0  = ((U[-1][0] - U[-1][1])*(U[-1][0] - U[-1][1])) in
    let top_m1_0    = ((U[-1][0] - U[-2][0])*(U[-1][0] - U[-2][0])) in
    let bottom_m1_0 = ((U[-1][0] - U[0][0])*(U[-1][0] - U[0][0])) in
    let g_m1_0      = rsqrt(epsilon + right_m1_0 + left_m1_0 + top_m1_0 + bottom_m1_0) in

    let left_p1_0   = ((U[1][0] - U[1][-1])*(U[1][0] - U[1][-1])) in
    let right_p1_0  = ((U[1][0] - U[1][1])*(U[1][0] - U[1][1])) in
    let top_p1_0    = ((U[1][0] - U[0][0])*(U[1][0] - U[0][0])) in
    let bottom_p1_0 = ((U[1][0] - U[2][0])*(U[1][0] - U[2][0])) in
    let g_p1_0      = rsqrt(epsilon + right_p1_0 + left_p1_0 + top_p1_0 + bottom_p1_0) in

    
    let left   = U[-1][0]*g_m1_0 in
    let right  = U[1][0]*g_p1_0 in
    let top    = U[0][-1]*g_0_m1 in
    let bottom = U[0][1]*g_0_p1 in

    (U[0][0] + DT*(right + left + top + bottom + gamma*F[0][0]*r)) /
      (1.0 + DT*(g_0_p1 + g_0_m1 + g_m1_0 + g_p1_0 + gamma))

    
#pragma overtile end

#endif

  double Stop = rtclock();

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";


  double GStencils = (Dim_0-2)*(Dim_1-2)*(double)TIME_STEPS/1e9/(Stop-Start);
  std::cout << "GStencils/sec: " << GStencils << "\n";

  
  delete [] U;
  delete [] F;

  return 0;
}
