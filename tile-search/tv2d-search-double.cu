#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <sys/time.h>
#include <iostream>

#ifndef PROBLEM_SIZE
#define PROBLEM_SIZE 6000
#endif

#ifndef TIME_STEPS
#define TIME_STEPS 100
#endif


inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

#define SQR(x) ((x)*(x))

int main() {

  const int Dim_0 = PROBLEM_SIZE;
  const int Dim_1 = PROBLEM_SIZE;
  
  double *V = new double[Dim_0*Dim_1];
  double *u = new double[Dim_0*Dim_1];
  double *f = new double[Dim_0*Dim_1];

  hipDeviceSynchronize();

  double Start = rtclock();

#pragma sdsl begin time_steps:TIME_STEPS TILE_SIZE_PARAMS

program tv2d is

  grid 2

  field u double inout
  field V double in
  field f double in


  u[1:1][1:1] = 
  let epi2    = 1e-8 in
  let alpha   = 1.001 in
  let c1      = u[0][0] * rsqrt(epi2 + SQR(u[1][0]-u[0][0]) + SQR(u[0][1]-u[0][0])) / V[0][0] in
  let c2      = u[0][0] * rsqrt(epi2 + SQR(u[0][0]-u[-1][0]) + SQR(u[-1][1]-u[-1][0])) / V[0][0] in
  let c3      = u[0][0] * rsqrt(epi2 + SQR(u[1][0]-u[0][0]) + SQR(u[0][1]-u[0][0])) / V[0][0] in
  let c4      = u[0][0] * rsqrt(epi2 + SQR(u[1][-1]-u[0][-1]) + SQR(u[0][0]-u[0][-1])) / V[0][0] in
       
    1.0 / (alpha+c1+c2+c3+c4)*(alpha*f[0][0]+c1*u[1][0]+c2*u[-1][0]+c3*u[0][1]+c4*u[0][-1])

#pragma sdsl end

  double Stop = rtclock();

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";

  double GStencils = (Dim_0-2)*(Dim_1-2)*(double)TIME_STEPS/1e9/(Stop-Start);
  std::cout << "GStencils/sec: " << GStencils << "\n";
  
  delete [] V;
  delete [] u;
  delete [] f;

  return 0;
}
