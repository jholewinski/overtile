
#include <sys/time.h>
#include <iostream>

#ifndef PROBLEM_SIZE
#define PROBLEM_SIZE 4000
#endif

#ifndef TIME_STEPS
#define TIME_STEPS 100
#endif


inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}





int main() {

  const int Dim_0 = PROBLEM_SIZE;
  const int Dim_1 = PROBLEM_SIZE;
  
  float *Ex = new float[Dim_0*Dim_1];
  float *Ey = new float[Dim_0*Dim_1];
  float *Hz = new float[Dim_0*Dim_1];

  hipDeviceSynchronize();

  double Start = rtclock();
  
#pragma overtile begin time_steps:TIME_STEPS TILE_SIZE_PARAMS

  program fdtd2d is
  grid 2
  field Ex float inout
  field Ey float inout
  field Hz float inout
    
    Ey[1:0][0:0] = Ey[0][0] - 0.5*(Hz[0][0] - Hz[-1][0])
    Ex[0:0][1:0] = Ex[0][0] - 0.5*(Hz[0][0] - Hz[0][-1])
    Hz[0:1][0:1] = Hz[0][0] - 0.7*(Ex[0][1] - Ex[0][0] + Ey[1][0] - Ey[0][0])
    
#pragma overtile end

  double Stop = rtclock();

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";

  double GStencils = (PROBLEM_SIZE-2)*(PROBLEM_SIZE-2)*(double)TIME_STEPS/1e9/(Stop-Start);
  
  std::cout << "GStencils/sec: " << GStencils << "\n";
  
  delete [] Ex;
  delete [] Ey;
  delete [] Hz;

  return 0;
}
