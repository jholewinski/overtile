#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <sys/time.h>
#include <iostream>

#ifndef PROBLEM_SIZE
#define PROBLEM_SIZE 300
#endif

#ifndef TIME_STEPS
#define TIME_STEPS 100
#endif


inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}


#define SQR(x) ((x)*(x))
  
int main() {

  const int Dim_0 = PROBLEM_SIZE;
  const int Dim_1 = PROBLEM_SIZE;
  const int Dim_2 = PROBLEM_SIZE;
  
  double *Image = new double[Dim_0*Dim_1*Dim_2];
  double *SumA = new double[Dim_0*Dim_1*Dim_2];


  hipDeviceSynchronize();

  double Start = rtclock();

#ifndef REF_TEST
  
#pragma overtile begin time_steps:TIME_STEPS TILE_SIZE_PARAMS

  program tv3d is
    
  grid 3

  field Image double inout
  field SumA double in

    Image[1:1][1:1] =
    let eps = 1e-8 in
    let alpha = 1.001 in

    let c1 = Image[0][0][0] * rsqrt(eps + SQR(Image[1][0][0] - Image[0][0][0]) + SQR(Image[0][1][0] - Image[0][0][0]) + SQR(Image[0][0][1] - Image[0][0][0])) / SumA[0][0][0] in
    let c2 = Image[0][0][0] * rsqrt(eps + SQR(Image[0][0][0] - Image[-1][0][0]) + SQR(Image[-1][1][0] - Image[-1][0][0]) + SQR(Image[-1][0][1] - Image[-1][0][0])) / SumA[0][0][0] in
    let c3 = Image[0][0][0] * rsqrt(eps + SQR(Image[1][-1][0] - Image[0][-1][0]) + SQR(Image[0][0][0] - Image[0][-1][0]) + SQR(Image[0][-1][1] - Image[0][-1][0])) / SumA[0][0][0] in
    let c4 = Image[0][0][0] * rsqrt(eps + SQR(Image[1][0][-1] - Image[0][0][-1]) + SQR(Image[0][1][-1] - Image[0][0][-1]) + SQR(Image[0][0][0] - Image[0][0][-1])) / SumA[0][0][0] in

    (alpha*Image[0][0][0] + c1*(Image[1][0][0] + Image[0][1][0] + Image[0][0][1]) + c2*Image[-1][0][0] + c3*Image[0][-1][0] + c4*Image[0][0][-1])/(alpha + 3.0*c1 + c2 + c3 + c4)

    
#pragma overtile end

#endif

  double Stop = rtclock();

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";


  double GStencils = (Dim_0-2)*(Dim_1-2)*(Dim_2-2)*(double)TIME_STEPS/1e9/(Stop-Start);
  std::cout << "GStencils/sec: " << GStencils << "\n";
  
  delete [] Image;
  delete [] SumA;

  return 0;
}
