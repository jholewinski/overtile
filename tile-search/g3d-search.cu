#include "hip/hip_runtime.h"

#include <sys/time.h>
#include <iostream>

#ifndef PROBLEM_SIZE
#define PROBLEM_SIZE 300
#endif

#ifndef TIME_STEPS
#define TIME_STEPS 100
#endif


inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}





int main() {

  const int Dim_0 = PROBLEM_SIZE;
  const int Dim_1 = PROBLEM_SIZE;
  const int Dim_2 = PROBLEM_SIZE;
  
  float *A = new float[Dim_0*Dim_1*Dim_2];

  hipDeviceSynchronize();

  double Start = rtclock();
  
#pragma overtile begin time_steps:TIME_STEPS TILE_SIZE_PARAMS

  program g3d is
  grid 3
  field A float inout

    A[1:1][1:1][1:1] = 
    
    let p0 = (A[0][0][0] - A[0][0][1]) * (A[0][0][0] - A[0][0][1]) in
    let p1 = (A[0][0][0] - A[0][0][-1]) * (A[0][0][0] - A[0][0][-1]) in
    let p2 = (A[0][0][0] - A[0][1][0]) * (A[0][0][0] - A[0][1][0]) in
    let p3 = (A[0][0][0] - A[0][-1][0]) * (A[0][0][0] - A[0][-1][0]) in
    let p4 = (A[0][0][0] - A[1][0][0]) * (A[0][0][0] - A[1][0][0]) in
    let p5 = (A[0][0][0] - A[-1][0][0]) * (A[0][0][0] - A[-1][0][0]) in
      A[0][0][0] + rsqrt(0.00001 + p0 + p1 + p2 + p3 + p4 + p5)

#pragma overtile end

  double Stop = rtclock();

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";

  double GStencils = (PROBLEM_SIZE-2)*(PROBLEM_SIZE-2)*(PROBLEM_SIZE-2)*(double)TIME_STEPS/1e9/(Stop-Start);
  
  std::cout << "GStencils/sec: " << GStencils << "\n";

  delete [] A;

  return 0;
}
