
#include <sys/time.h>
#include <iostream>

#ifndef PROBLEM_SIZE
#define PROBLEM_SIZE 200
#endif

#ifndef TIME_STEPS
#define TIME_STEPS 50
#endif


inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}





int main() {

  const int Dim_0 = PROBLEM_SIZE;
  const int Dim_1 = PROBLEM_SIZE;
  const int Dim_2 = PROBLEM_SIZE;
  
  float *A = new float[Dim_0*Dim_1*Dim_2];

  hipDeviceSynchronize();

  double Start = rtclock();
  
#pragma overtile begin time_steps:TIME_STEPS TILE_SIZE_PARAMS

  program j3d is
  grid 3
  field A float inout
    
    A[1:1][1:1][1:1] = 0.143 * (A[-1][0][0] + A[0][0][0] + A[1][0][0] + A[0][-1][0] + A[0][1][0] + A[0][0][-1] + A[0][0][1])

#pragma overtile end

  double Stop = rtclock();

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";

  double GStencils = (PROBLEM_SIZE-2)*(PROBLEM_SIZE-2)*(PROBLEM_SIZE-2)*(double)TIME_STEPS/1e9/(Stop-Start);
  
  std::cout << "GStencils/sec: " << GStencils << "\n";

  delete [] A;

  return 0;
}
