#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <sys/time.h>
#include <iostream>

#ifndef PROBLEM_SIZE
#define PROBLEM_SIZE 6000
#endif

#ifndef TIME_STEPS
#define TIME_STEPS 100
#endif


inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}


#define SQR(x) ((x)*(x))


int main() {

  const int Dim_0 = PROBLEM_SIZE;
  const int Dim_1 = PROBLEM_SIZE;
  
  float *U = new float[Dim_0*Dim_1];
  float *F = new float[Dim_0*Dim_1];




  hipDeviceSynchronize();

  double Start = rtclock();

#ifndef REF_TEST
  
#pragma sdsl begin time_steps:TIME_STEPS TILE_SIZE_PARAMS

  program rician2d is
  grid 2
  field U float inout
  field F float in

  U[1:1][1:1] =

    let DT      = 5.0 in
    let sigma   = 1.00001 in
    let sigma2  = sigma*sigma in
    let lambda  = 1.00001 in
    let gamma   = lambda/sigma2 in
    let r_inner = U[0][0]*F[0][0]/sigma2 in
    let r       = (r_inner*(2.38944 + r_inner*(0.950037+r_inner))) / (4.65314 + r_inner*(2.57541 + r_inner*(1.48937 + r_inner))) in
    let epsilon = 1.0e-20 in


    let left_0_m1   = ((U[0][-1] - U[0][-2])*(U[0][-1] - U[0][-2])) in
    let right_0_m1  = ((U[0][-1] - U[0][0])*(U[0][-1] - U[0][0])) in
    let top_0_m1    = ((U[0][-1] - U[-1][-1])*(U[0][-1] - U[-1][-1])) in
    let bottom_0_m1 = ((U[0][-1] - U[1][-1])*(U[0][-1] - U[1][-1])) in
    let g_0_m1      = rsqrtf(epsilon + right_0_m1 + left_0_m1 + top_0_m1 + bottom_0_m1) in
    
    let left_0_p1   = ((U[0][1] - U[0][0])*(U[0][1] - U[0][0])) in
    let right_0_p1  = ((U[0][1] - U[0][2])*(U[0][1] - U[0][2])) in
    let top_0_p1    = ((U[0][1] - U[-1][1])*(U[0][1] - U[-1][1])) in
    let bottom_0_p1 = ((U[0][1] - U[1][1])*(U[0][1] - U[1][1])) in
    let g_0_p1      = rsqrtf(epsilon + right_0_p1 + left_0_p1 + top_0_p1 + bottom_0_p1) in

    let left_m1_0   = ((U[-1][0] - U[-1][-1])*(U[-1][0] - U[1][-1])) in
    let right_m1_0  = ((U[-1][0] - U[-1][1])*(U[-1][0] - U[-1][1])) in
    let top_m1_0    = ((U[-1][0] - U[-2][0])*(U[-1][0] - U[-2][0])) in
    let bottom_m1_0 = ((U[-1][0] - U[0][0])*(U[-1][0] - U[0][0])) in
    let g_m1_0      = rsqrtf(epsilon + right_m1_0 + left_m1_0 + top_m1_0 + bottom_m1_0) in

    let left_p1_0   = ((U[1][0] - U[1][-1])*(U[1][0] - U[1][-1])) in
    let right_p1_0  = ((U[1][0] - U[1][1])*(U[1][0] - U[1][1])) in
    let top_p1_0    = ((U[1][0] - U[0][0])*(U[1][0] - U[0][0])) in
    let bottom_p1_0 = ((U[1][0] - U[2][0])*(U[1][0] - U[2][0])) in
    let g_p1_0      = rsqrtf(epsilon + right_p1_0 + left_p1_0 + top_p1_0 + bottom_p1_0) in

    
    let left   = U[-1][0]*g_m1_0 in
    let right  = U[1][0]*g_p1_0 in
    let top    = U[0][-1]*g_0_m1 in
    let bottom = U[0][1]*g_0_p1 in

    (U[0][0] + DT*(right + left + top + bottom + gamma*F[0][0]*r)) /
      (1.0 + DT*(g_0_p1 + g_0_m1 + g_m1_0 + g_p1_0 + gamma))

    
#pragma sdsl end

#endif

  double Stop = rtclock();

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";


  double GStencils = (Dim_0-2)*(Dim_1-2)*(double)TIME_STEPS/1e9/(Stop-Start);
  std::cout << "GStencils/sec: " << GStencils << "\n";

  
  delete [] U;
  delete [] F;

  return 0;
}
