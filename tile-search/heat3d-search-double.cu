
#include <sys/time.h>
#include <iostream>

#ifndef PROBLEM_SIZE
#define PROBLEM_SIZE 300
#endif

#ifndef TIME_STEPS
#define TIME_STEPS 100
#endif


inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}





int main() {

  const int Dim_0 = PROBLEM_SIZE;
  const int Dim_1 = PROBLEM_SIZE;
  const int Dim_2 = PROBLEM_SIZE;
  
  double *A = new double[Dim_0*Dim_1*Dim_2];

  hipDeviceSynchronize();

  double Start = rtclock();

  const double      C1 = 2.0f;
  const double C2 = 0.125f;
  
#pragma overtile begin time_steps:TIME_STEPS TILE_SIZE_PARAMS

  program heat3d is
  grid 3
  field A double inout
    
    A[1:1][1:1][1:1] = 
      0.125 * (A[1][0][0] - 2.0 * A[0][0][0] + A[-1][0][0])
    + 0.125 * (A[0][1][0] - 2.0 * A[0][0][0] + A[0][-1][0])
    + 0.125 * (A[0][0][1] - 2.0 * A[0][0][0] + A[0][0][-1])
    + A[0][0][0]
#pragma overtile end

   double Stop = rtclock();

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";

  double GStencils = (PROBLEM_SIZE-2)*(PROBLEM_SIZE-2)*(PROBLEM_SIZE-2)*(double)TIME_STEPS/1e9/(Stop-Start);
  
  std::cout << "GStencils/sec: " << GStencils << "\n";

  delete [] A;

  return 0;
}
