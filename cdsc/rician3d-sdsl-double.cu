#include "hip/hip_runtime.h"
/*
 * Implementation of Rician 3D pipeline stage using embedded DSL.
 */

#include <sys/time.h>
#include <iostream>

namespace {
const size_t MaxIter     = 1000;
const size_t IterPerCall = 4;
const double  Tolerance   = 2e-3f;
}


int RunRician3D(double *U, double *F, size_t Dim_0, size_t Dim_1, size_t Dim_2) {

  // G is a local array
  double *G = new double[Dim_0*Dim_1*Dim_2];
  memset(G, 0, sizeof(double)*Dim_0*Dim_1*Dim_2);

  // Zero out U
  memcpy(U, F, sizeof(double)*Dim_0*Dim_1*Dim_2);

  // Create "old" U
  double *OldU = new double[Dim_0*Dim_1*Dim_2];

  int  Iter;
  bool Converged = false;
  
  for (Iter = 0; Iter < MaxIter && !Converged; Iter += IterPerCall) {

    // Take copy of existing U
    memcpy(OldU, U, sizeof(double)*Dim_0*Dim_1*Dim_2);
    
    // Run kernel
#pragma sdsl begin time_steps:IterPerCall block:8,8,8 tile:1,1,1 time:1
int Nx;
int Ny;
int Nz;

double DT = 5.0;
double EPSILON = 1.0E-20;

double sigma = 0.05;
double lambda = 0.065;
double sigma2 = sigma*sigma;
double gamma = lambda/sigma2;

grid g[Nz][Ny][Nx];

double griddata U on g at 0,1;
double griddata G on g at 0,1;
double griddata F on g at 0;


pointfunction approx_g(u,g) {
  [1]g[0][0][0] = 1.0 / sqrt(EPSILON +
                         ([0]u[0][0][0] - [0]u[ 0][ 1][ 0])*([0]u[0][0][0] - [0]u[ 0][ 1][ 0]) +
                         ([0]u[0][0][0] - [0]u[ 0][-1][ 0])*([0]u[0][0][0] - [0]u[ 0][-1][ 0]) +
                         ([0]u[0][0][0] - [0]u[ 0][ 0][ 1])*([0]u[0][0][0] - [0]u[ 0][ 0][ 1]) +
                         ([0]u[0][0][0] - [0]u[ 0][ 0][-1])*([0]u[0][0][0] - [0]u[ 0][ 0][-1]) +
                         ([0]u[0][0][0] - [0]u[ 1][ 0][ 0])*([0]u[0][0][0] - [0]u[ 1][ 0][ 0]) +
                         ([0]u[0][0][0] - [0]u[-1][ 0][ 0])*([0]u[0][0][0] - [0]u[-1][ 0][ 0]) );
}

pointfunction update_u(u,g,f) {
  double tmp = [0]u[0][0][0]*[0]f[0][0][0]/sigma2;
  double r =   ( tmp*(2.38944 + tmp*(0.950037 + tmp)) )
                  / ( 4.65314 + tmp*(2.57541 + tmp*(1.48937 + tmp)) );

  [1]u[0][0][0] =   ([0]u[0][0][0] +
                     DT*([0]u[ 0][ 1][ 0]*[0]g[ 0][ 1][ 0] +
                         [0]u[ 0][-1][ 0]*[0]g[ 0][-1][ 0] +
                         [0]u[ 0][ 0][ 1]*[0]g[ 0][ 0][ 1] +
                         [0]u[ 0][ 0][-1]*[0]g[ 0][ 0][-1] +
                         [0]u[ 1][ 0][ 0]*[0]g[ 1][ 0][ 0] +
                         [0]u[-1][ 0][ 0]*[0]g[-1][ 0][ 0] +
                         gamma*[0]f[0][0][0]*r))
                   /
                    (1.0 + DT*[1]g[ 0][ 1][ 0] +
                              [1]g[ 0][-1][ 0] +
                              [1]g[ 0][ 0][ 1] +
                              [1]g[ 0][ 0][-1] +
                              [1]g[ 1][ 0][ 0] +
                              [1]g[-1][ 0][ 0] +
                              gamma);
}

iterate 100 {
  stencil gs {
    [1:Nz-1][1:Ny-1][1:Nx-1] : approx_g(U,G); 
  }
  stencil us {
    [1:Nz-1][1:Ny-1][1:Nx-1] : update_u(U, G, F); 
  }
} check (0) every 10 iterations    
#pragma sdsl end

    // Check for convergence
    Converged = true;
    
    for (int i = 0; i < Dim_0*Dim_1*Dim_2; ++i) {
      if (std::abs(OldU[i] - U[i]) > Tolerance) {
        Converged = false;
        break;
      }
    }
  }

  
  // Clean-up
  delete [] G;

  
  if (Iter >= MaxIter) {
    return 0;                   // Did not converge!
  }

#ifdef STANDALONE_DRIVER
  std::cout << "Converged in <= " << Iter << " iterations\n";
#endif

  return Iter;
}


#ifdef STANDALONE_DRIVER

inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}


int main() {

  const int Dim_0     = 200;
  const int Dim_1     = 200;
  const int Dim_2     = 200;
  const int TimeSteps = 100;
  
  double *U = new double[Dim_0*Dim_1*Dim_2];
  double *F = new double[Dim_0*Dim_1*Dim_2];

  srand(time(NULL));
  
  for (int i = 0; i < Dim_0*Dim_1*Dim_2; ++i) {
    F[i] = (double)rand() / (double)(RAND_MAX+1.0f) * 10.0f;
  }
  

  hipDeviceSynchronize();

  double Start = rtclock();
  
  int Ret = RunRician3D(U, F, Dim_0, Dim_1, Dim_2);

  double Stop = rtclock();

  if (Ret == 0) {
    std::cout << "Did not converge!\n";
    return 0;
  }

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";

  double GStencils = (Dim_0-2)*(Dim_1-2)*(Dim_2-2)*(double)Ret/1e9/(Stop-Start);

  std::cout << "GStencils/sec: " << GStencils << "\n";

  delete [] U;
  delete [] F;

  return 0;
}

#endif
