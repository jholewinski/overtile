#include "hip/hip_runtime.h"
/*
 * Implementation of Rician 3D pipeline stage using embedded Stencil Specification.
 */

#include <sys/time.h>
#include <iostream>

namespace {
const size_t MaxIter     = 1000;
const size_t IterPerCall = 4;
const double Tolerance   = 2e-3;
}


int RunRician3D(double *U, double *F, size_t Dim_0, size_t Dim_1, size_t Dim_2) {

  // G is a local array
  double *G = new double[Dim_0*Dim_1*Dim_2];
  memset(G, 0, sizeof(double)*Dim_0*Dim_1*Dim_2);

  // Zero out U
  memcpy(U, F, sizeof(double)*Dim_0*Dim_1*Dim_2);

  // Create "old" U
  double *OldU = new double[Dim_0*Dim_1*Dim_2];

  int  Iter;
  bool Converged = false;
  
  for (Iter = 0; Iter < MaxIter && !Converged; Iter += IterPerCall) {

    // Take copy of existing U
    memcpy(OldU, U, sizeof(double)*Dim_0*Dim_1*Dim_2);
    
    // Run kernel
#pragma sdsl begin time_steps:IterPerCall block:8,8,8 tile:1,1,1 time:1

    program rician3d is
      grid 3
      field G double inout
      field U double inout
      field F double in

      F[0:0][0:0][0:0] = F[0][0][0]
    
      G[1:1][1:1][1:1] = 
    
      let left    = ((U[0][0][0] - U[0][0][-1])*(U[0][0][0] - U[0][0][-1])) in
      let right   = ((U[0][0][0] - U[0][0][1])*(U[0][0][0] - U[0][0][1])) in
      let top     = ((U[0][0][0] - U[0][-1][0])*(U[0][0][0] - U[0][-1][0])) in
      let bottom  = ((U[0][0][0] - U[0][1][0])*(U[0][0][0] - U[0][1][0])) in
      let back    = ((U[0][0][0] - U[-1][0][0])*(U[0][0][0] - U[-1][0][0])) in
      let front   = ((U[0][0][0] - U[1][0][0])*(U[0][0][0] - U[1][0][0])) in
      let epsilon = 1.0e-20 in

      rsqrt(epsilon + right + left + top + bottom + back + front)

      U[1:1][1:1][1:1] = 

      let DT      = 5.0 in
      let sigma   = 0.05 in
      let sigma2  = sigma*sigma in
      let lambda  = 0.065 in
      let gamma   = lambda/sigma2 in
      let r_inner = U[0][0][0]*F[0][0][0]/sigma2 in
      let r       = (r_inner*(2.38944 + r_inner*(0.950037+r_inner))) / (4.65314 + r_inner*(2.57541 + r_inner*(1.48937 + r_inner))) in

      let left   = U[0][-1][0]*G[0][-1][0] in
      let right  = U[0][1][0]*G[0][1][0] in
      let top    = U[0][0][-1]*G[0][0][-1] in
      let bottom = U[0][0][1]*G[0][0][1] in
      let back   = U[-1][0][0]*G[-1][0][0] in
      let front  = U[1][0][0]*G[1][0][0] in

      (U[0][0][0] + DT*(right + left + top + bottom + back + front + gamma*F[0][0][0]*r)) /
      (1.0 + DT*(G[0][0][1] + G[0][0][-1] + G[0][-1][0] + G[0][1][0] + G[-1][0][0] + G[1][0][0] + gamma))

    
#pragma sdsl end

    // Check for convergence
    Converged = true;
    
    for (int i = 0; i < Dim_0*Dim_1*Dim_2; ++i) {
      if (std::abs(OldU[i] - U[i]) > Tolerance) {
        Converged = false;
        break;
      }
    }
  }

  
  // Clean-up
  delete [] G;

  
  if (Iter >= MaxIter) {
    return 0;                   // Did not converge!
  }

#ifdef STANDALONE_DRIVER
  std::cout << "Converged in <= " << Iter << " iterations\n";
#endif

  return Iter;
}


#ifdef STANDALONE_DRIVER

inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}


int main() {

  const int Dim_0     = 200;
  const int Dim_1     = 200;
  const int Dim_2     = 200;
  const int TimeSteps = 100;
  
  double *U = new double[Dim_0*Dim_1*Dim_2];
  double *F = new double[Dim_0*Dim_1*Dim_2];

  srand(time(NULL));
  
  for (int i = 0; i < Dim_0*Dim_1*Dim_2; ++i) {
    F[i] = (double)rand() / (double)(RAND_MAX+1.0f) * 10.0;
  }
  

  hipDeviceSynchronize();

  double Start = rtclock();
  
  int Ret = RunRician3D(U, F, Dim_0, Dim_1, Dim_2);

  double Stop = rtclock();

  if (Ret == 0) {
    std::cout << "Did not converge!\n";
    return 0;
  }

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";

  double GStencils = (Dim_0-2)*(Dim_1-2)*(Dim_2-2)*(double)Ret/1e9/(Stop-Start);

  std::cout << "GStencils/sec: " << GStencils << "\n";

  delete [] U;
  delete [] F;

  return 0;
}

#endif
