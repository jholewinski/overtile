#include "hip/hip_runtime.h"
/*
 * Implementation of Segmentation 3D pipeline stage using embedded DSL.
 */

#include <sys/time.h>
#include <iostream>

namespace {
const int MaxIter       = 100;
const int IterPerKernel = 10;
}

#define REF3D(i,j,k) ((i)*Dim_0*Dim_1+(j)*Dim_0+(k))

#define		MDP(a, b)	(2.0 * a / (b * b))
#define		MU	(0.18 * 255 * 255)
#define   DX  1.0
#define   DY  1.0
#define   DZ  1.0
#define		SUM	((MDP(MU, DX)) + (MDP(MU, DY)) + (MDP(MU, DZ)))
#define		DT1	((SUM > 1.0e-15) ? (0.9 / SUM) : (100.0)) /* Compute the Courant number for Active Contours CV */
#define		DT2	(10e7 * DT1)


int RunSegmentation3D(double *u0, double *phi, size_t Dim_0, size_t Dim_1, size_t Dim_2) {

  double num1, num2, den1, den2, c1, c2;

  
  num1 = 0.0;
  num2 = 0.0;
  den1 = 0.0;
  den2 = 0.0;

  for (int i = 0; i < MaxIter; i += IterPerKernel) {

    for(int i = 0; i < Dim_2; i++) {
      for(int j = 0; j < Dim_1; j++) {
        for(int k = 0; k < Dim_0; k++) {
          num1 = (phi[REF3D(i,j,k)] < 0) ? num1 + u0[REF3D(i,j,k)] : (num1);
          den1 = (phi[REF3D(i,j,k)] < 0) ? den1 + 1 : (den1);
          num2 = (phi[REF3D(i,j,k)] > 0) ? num2 + u0[REF3D(i,j,k)] : (num2);
          den2 = (phi[REF3D(i,j,k)] > 0) ? den2 + 1 : (den2);
        }
      }
    }

    c1 = num1 / den1;
    c2 = num2 / den2;

    double DT = DT2;

#pragma overtile begin time_steps:IterPerKernel block:8,8,8 tile:1,1,1 time:1
int                                   Nx;
int                                   Ny;
int                                   Nz;

double c1;
double c2;
double DT;

double DX = 1.0;
double DY = 1.0;
double DZ = 1.0;

double DX2 = DX * 2.0;
double DY2 = DY * 2.0;
double DZ2 = DZ * 2.0;

double MU = 0.18 * 255.0 * 255.0;
double LAMBDA1 = 1.0;
double LAMBDA2 = 1.0;
double EPSILON = 10.0E-7;
double EPSILON_SQRD = EPSILON*EPSILON;


grid g[Nz][Ny][Nx];

double griddata u0  on g at 0,1;
double griddata phi on g at 0,1;

pointfunction calc_phi(p,u) {
  double Dx_p = ([0]p[1][0][0] - [0]p[ 0][ 0][ 0])/DX;
  double Dx_m = ([0]p[0][0][0] - [0]p[-1][ 0][ 0])/DX;
  double Dy_p = ([0]p[0][1][0] - [0]p[ 0][ 0][ 0])/DY;
  double Dy_m = ([0]p[0][0][0] - [0]p[ 0][-1][ 0])/DY;
  double Dz_p = ([0]p[0][0][1] - [0]p[ 0][ 0][ 0])/DZ;
  double Dz_m = ([0]p[0][0][0] - [0]p[ 0][ 0][-1])/DZ;

  double Dx_0 = ([0]p[1][0][0] - [0]p[-1][ 0][ 0])/DX2;
  double Dy_0 = ([0]p[0][1][0] - [0]p[ 0][-1][ 0])/DY2;
  double Dz_0 = ([0]p[0][0][1] - [0]p[ 0][ 0][-1])/DZ2;

  double C1x  =  1.0 / sqrt( Dx_p*Dx_p + Dy_0*Dy_0 + Dz_0*Dz_0  + EPSILON_SQRD);
  double C2x  =  1.0 / sqrt( Dx_m*Dx_m + Dy_0*Dy_0 + Dz_0*Dz_0  + EPSILON_SQRD);
  double C3y  =  1.0 / sqrt( Dx_0*Dx_0 + Dy_p*Dy_p + Dz_0*Dz_0  + EPSILON_SQRD);
  double C4y  =  1.0 / sqrt( Dx_0*Dx_0 + Dy_m*Dy_m + Dz_0*Dz_0  + EPSILON_SQRD);
  double C5z  =  1.0 / sqrt( Dx_0*Dx_0 + Dy_0*Dy_0 + Dz_p*Dz_p  + EPSILON_SQRD);
  double C6z  =  1.0 / sqrt( Dx_0*Dx_0 + Dy_0*Dy_0 + Dz_m*Dz_m  + EPSILON_SQRD);

  double Grad = sqrt(Dx_0*Dx_0 + Dy_0*Dy_0 + Dz_0*Dz_0);

  double MM  =  (DT/(DX*DY)) * Grad * MU;
  double CC  =  1 + MM*(C1x + C2x + C3y + C4y + C5z + C6z);

  double C1x_2x  =  C1x*[0]p[1][0][0] + C2x*[0]p[-1][ 0][ 0];
  double C3y_4y  =  C3y*[0]p[0][1][0] + C4y*[0]p[ 0][-1][ 0];
  double C5z_6z  =  C5z*[0]p[0][0][1] + C6z*[0]p[ 0][ 0][-1];

  [1]p[0][0][0] = (1.0 / CC) * ( [0]p[0][0][0] + MM*( C1x_2x + C3y_4y + C5z_6z )
                              + (DT*Grad)* ( (LAMBDA1*([0]u[0][0][0] - c1)*([0]u[0][0][0] - c1))
                              -              (LAMBDA2*([0]u[0][0][0] - c2)*([0]u[0][0][0] - c2))));
}


iterate 100 {
  stencil segi {
    [1:Nz-2][1:Ny-2][1:Nx-2] : calc_phi(phi, u0);
    
  }
} check (0 == 0) every 100 iterations

#pragma overtile end
    }

    /*
          [1:Nz-1][1:Ny-1][0     ] : [1]phi[0][0]{0   } = [0]phi[0][0]{1   };
    [1:Nz-1][1:Ny-1][Nx-1  ] : [1]phi[0][0]{Nx-1} = [0]phi[0][0]{Nx-2};
    
    [1:Nz-1][0     ][1:Nx-1] : [1]phi[0]{0   }[0] = [0]phi[0]{1   }[0];
    [1:Nz-1][Ny-1  ][1:Nx-1] : [1]phi[0]{Ny-1}[0] = [0]phi[0]{Ny-2}[0];
    
    [0     ][1:Ny-1][1:Nx-1] : [1]phi{0   }[0][0] = [0]phi{1   }[0][0];
    [Nz-1  ][1:Ny-1][1:Nx-1] : [1]phi{Nz-1}[0][0] = [0]phi{Ny-2}[0][0];
    */

  return 0;
}


#ifdef STANDALONE_DRIVER

inline double rtclock() {
  struct timeval Tp;
  gettimeofday(&Tp, NULL);
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}


int main() {

  const int Dim_0     = 200;
  const int Dim_1     = 200;
  const int Dim_2     = 200;
  const int TimeSteps = 100;
  
  double *u0 = new double[Dim_0*Dim_1*Dim_2];
  double *phi = new double[Dim_0*Dim_1*Dim_2];

  srand(time(NULL));
  
  for (int i = 0; i < Dim_0*Dim_1*Dim_2; ++i) {
    u0[i] = (double)rand() / (double)(RAND_MAX+1.0f) * 10.0f;
    phi[i] = (double)rand() / (double)(RAND_MAX+1.0f) * 10.0f;
  }
  

  hipDeviceSynchronize();

  double Start = rtclock();
  
  int Ret = RunSegmentation3D(u0, phi, Dim_0, Dim_1, Dim_2);

  double Stop = rtclock();

  std::cout << "CPU Elapsed: " << (Stop-Start) << "\n";

  double GStencils = (Dim_0-2)*(Dim_1-2)*(Dim_2-2)*(double)TimeSteps/1e9/(Stop-Start);

  std::cout << "GStencils/sec: " << GStencils << "\n";

  delete [] u0;
  delete [] phi;

  return 0;
}

#endif
